#include "hip/hip_runtime.h"
#include "quantize.cuh"
#include <cstdint>

static __global__ void quantize_q8_1(const float * __restrict__ x, void * __restrict__ vy, const int64_t kx, const int64_t kx0_padded) {
    const int64_t ix0 = (int64_t)blockDim.x*blockIdx.x + threadIdx.x;

    if (ix0 >= kx0_padded) {
        return;
    }

    const int64_t ix1 = blockIdx.y;

    const int64_t i_padded = ix1*kx0_padded + ix0;

    block_q8_1 * y = (block_q8_1 *) vy;

    const int64_t ib = i_padded / QK8_1; // block index
    const int64_t iqs = i_padded % QK8_1; // quant index

    const float xi = ix0 < kx ? x[ix1*kx + ix0] : 0.0f;
    float amax = fabsf(xi);
    float sum = xi;

    amax = warp_reduce_max(amax);
    sum = warp_reduce_sum(sum);

    const float d = amax / 127;
    const int8_t q = amax == 0.0f ? 0 : roundf(xi / d);

    y[ib].qs[iqs] = q;

    if (iqs > 0) {
        return;
    }

    reinterpret_cast<half&>(y[ib].ds.x) = d;
    reinterpret_cast<half&>(y[ib].ds.y) = sum;
}

template <bool need_sum>
static __global__ void quantize_mmq_q8_1(
    const float * __restrict__ x, void * __restrict__ vy, const int64_t kx0, const int64_t kx1, const int64_t kx0_padded) {

    const int64_t ix0 = ((int64_t)blockDim.x*blockIdx.x + threadIdx.x)*4;

    if (ix0 >= kx0_padded) {
        return;
    }

    const float4 * x4 = (const float4 *) x;

    const int64_t ix1 = kx1*blockIdx.z + blockIdx.y;

    block_q8_1_mmq * y = (block_q8_1_mmq *) vy;

    const int64_t ib0 = blockIdx.z*((int64_t)gridDim.y*gridDim.x*blockDim.x/QK8_1); // first block of channel
    const int64_t ib  = ib0 + (ix0 / (4*QK8_1))*kx1 + blockIdx.y;                   // block index in channel
    const int64_t iqs = ix0 % (4*QK8_1);                                            // quant index in block

    const float4 xi = ix0 < kx0 ? x4[(ix1*kx0 + ix0)/4] : make_float4(0.0f, 0.0f, 0.0f, 0.0f);
    float amax = fabsf(xi.x);
    amax = fmaxf(amax, fabsf(xi.y));
    amax = fmaxf(amax, fabsf(xi.z));
    amax = fmaxf(amax, fabsf(xi.w));

#pragma unroll
    for (int mask = 4; mask > 0; mask >>= 1) {
        amax = fmaxf(amax, __shfl_xor_sync(0xFFFFFFFF, amax, mask, WARP_SIZE));
    }

    float sum;
    if (need_sum) {
        sum = xi.x + xi.y + xi.z + xi.w;
#pragma unroll
        for (int mask = 4; mask > 0; mask >>= 1) {
            sum += __shfl_xor_sync(0xFFFFFFFF, sum, mask, WARP_SIZE);
        }
    }

    const float d_inv = 127.0f / amax;
    char4 q;
    q.x = roundf(xi.x*d_inv);
    q.y = roundf(xi.y*d_inv);
    q.z = roundf(xi.z*d_inv);
    q.w = roundf(xi.w*d_inv);

    char4 * yqs4 = (char4 *) y[ib].qs;
    yqs4[iqs/4] = q;

    if (iqs % QI8_1 != 0) {
        return;
    }

    const float d = 1.0f / d_inv;

    if (need_sum) {
        y[ib].ds[iqs/QK8_1] = make_half2(d, sum);
    } else {
        ((float *) y[ib].ds)[iqs/QK8_1] = d;
    }
}

void quantize_row_q8_1_cuda(
    const float * x, void * vy, const int64_t kx0, const int64_t kx1, const int64_t channels,
    const int64_t kx0_padded, const ggml_type type_x, hipStream_t stream) {

    GGML_ASSERT(kx0_padded % QK8_1 == 0);

    const int64_t block_num_x = (kx0_padded + CUDA_QUANTIZE_BLOCK_SIZE - 1) / CUDA_QUANTIZE_BLOCK_SIZE;
    const dim3 num_blocks(block_num_x, kx1*channels, 1);
    const dim3 block_size(CUDA_QUANTIZE_BLOCK_SIZE, 1, 1);
    quantize_q8_1<<<num_blocks, block_size, 0, stream>>>(x, vy, kx0, kx0_padded);

    GGML_UNUSED(type_x);
}

void quantize_mmq_q8_1_cuda(
    const float * x, void * vy, const int64_t kx0, const int64_t kx1, const int64_t channels,
    const int64_t kx0_padded, const ggml_type type_x, hipStream_t stream) {

    GGML_ASSERT(kx0_padded % (4*QK8_1) == 0);

    const int64_t block_num_x = (kx0_padded + 4*CUDA_QUANTIZE_BLOCK_SIZE - 1) / (4*CUDA_QUANTIZE_BLOCK_SIZE);
    const dim3 num_blocks(block_num_x, kx1, channels);
    const dim3 block_size(CUDA_QUANTIZE_BLOCK_SIZE, 1, 1);
    if (mmq_need_sum(type_x)) {
        quantize_mmq_q8_1<true><<<num_blocks, block_size, 0, stream>>>(x, vy, kx0, kx1, kx0_padded);
    } else {
        quantize_mmq_q8_1<false><<<num_blocks, block_size, 0, stream>>>(x, vy, kx0, kx1, kx0_padded);
    }
}
