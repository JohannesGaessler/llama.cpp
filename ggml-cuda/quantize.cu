#include "hip/hip_runtime.h"
#include "quantize.cuh"
#include <cstdint>

static __global__ void quantize_q8_1(const float * __restrict__ x, void * __restrict__ vy, const int64_t kx, const int64_t kx0_padded) {
    const int64_t ix0 = (int64_t)blockDim.x*blockIdx.x + threadIdx.x;

    if (ix0 >= kx0_padded) {
        return;
    }

    const int64_t ix1 = (int64_t)blockDim.y*blockIdx.y + threadIdx.y;

    const int64_t i_padded = (int64_t)ix1*kx0_padded + ix0;

    block_q8_1 * y = (block_q8_1 *) vy;

    const int64_t ib = i_padded / QK8_1; // block index
    const int64_t iqs = i_padded % QK8_1; // quant index

    const float xi = ix0 < kx ? x[ix1*kx + ix0] : 0.0f;
    float amax = fabsf(xi);
    float sum = xi;

    amax = warp_reduce_max(amax);
    sum = warp_reduce_sum(sum);

    const float d = amax / 127;
    const int8_t q = amax == 0.0f ? 0 : roundf(xi / d);

    y[ib].qs[iqs] = q;

    if (iqs > 0) {
        return;
    }

    reinterpret_cast<half&>(y[ib].ds.x) = d;
    reinterpret_cast<half&>(y[ib].ds.y) = sum;
}

static __global__ void quantize_mmq_q8_1(
    const float * __restrict__ x, void * __restrict__ vy, const int64_t kx0, const int64_t kx1, const int64_t kx0_padded) {

    const int64_t ix0 = (int64_t)blockDim.x*blockIdx.x + threadIdx.x;

    if (ix0 >= kx0_padded) {
        return;
    }

    const int64_t ix1 = (int64_t)blockDim.y*blockIdx.y + threadIdx.y;

    block_q8_1_mmq * y = (block_q8_1_mmq *) vy;

    const int64_t ib = (ix0 / (4*QK8_1))*kx1 + ix1; // block index
    const int64_t iqs = ix0 % (4*QK8_1);            // quant index

    const float xi = ix0 < kx0 ? x[ix1*kx0 + ix0] : 0.0f;
    float amax = fabsf(xi);
    float sum = xi;

    amax = warp_reduce_max(amax);
    sum = warp_reduce_sum(sum);

    const float d = amax / 127;
    const int8_t q = amax == 0.0f ? 0 : roundf(xi / d);

    y[ib].qs[iqs] = q;

    if (iqs % QK8_1 != 0) {
        return;
    }

    ((float *) y[ib].ds)[iqs/QK8_1] = d;
    // reinterpret_cast<half&>(y[ib].ds[iqs/QK8_1].x) = d;
    // reinterpret_cast<half&>(y[ib].ds[iqs/QK8_1].y) = sum;
}

void quantize_row_q8_1_cuda(
    const float * x, void * vy, const int64_t kx0, const int64_t kx1, const int64_t kx0_padded, hipStream_t stream) {

    GGML_ASSERT(kx0_padded % QK8_1 == 0);

    const int64_t block_num_x = (kx0_padded + CUDA_QUANTIZE_BLOCK_SIZE - 1) / CUDA_QUANTIZE_BLOCK_SIZE;
    const dim3 num_blocks(block_num_x, kx1, 1);
    const dim3 block_size(CUDA_QUANTIZE_BLOCK_SIZE, 1, 1);
    quantize_q8_1<<<num_blocks, block_size, 0, stream>>>(x, vy, kx0, kx0_padded);
}

void quantize_mmq_q8_1_cuda(
    const float * x, void * vy, const int64_t kx0, const int64_t kx1, const int64_t kx0_padded, hipStream_t stream) {

    GGML_ASSERT(kx0_padded % (4*QK8_1) == 0);

    const int64_t block_num_x = (kx0_padded + CUDA_QUANTIZE_BLOCK_SIZE - 1) / CUDA_QUANTIZE_BLOCK_SIZE;
    const dim3 num_blocks(block_num_x, kx1, 1);
    const dim3 block_size(CUDA_QUANTIZE_BLOCK_SIZE, 1, 1);
    quantize_mmq_q8_1<<<num_blocks, block_size, 0, stream>>>(x, vy, kx0, kx1, kx0_padded);
}
