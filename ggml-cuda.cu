#include "hip/hip_runtime.h"
#include <cstddef>
#include <cstdint>
#include <stdint.h>
#include <stdio.h>
#include <atomic>

#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hip/hip_fp16.h>

#include "ggml-cuda.h"
#include "ggml.h"

static_assert(sizeof(half) == sizeof(ggml_fp16_t), "wrong fp16 size");

#define CUDA_CHECK(err)                                                                 \
    do {                                                                                \
        hipError_t err_ = (err);                                                       \
        if (err_ != hipSuccess) {                                                      \
            fprintf(stderr, "CUDA error %d at %s:%d: %s\n", err_, __FILE__, __LINE__,   \
                hipGetErrorString(err_));                                              \
            exit(1);                                                                    \
        }                                                                               \
    } while (0)

#define CUBLAS_CHECK(err)                                                               \
    do {                                                                                \
        hipblasStatus_t err_ = (err);                                                    \
        if (err_ != HIPBLAS_STATUS_SUCCESS) {                                            \
            fprintf(stderr, "cuBLAS error %d at %s:%d\n", err_, __FILE__, __LINE__);    \
            exit(1);                                                                    \
        }                                                                               \
    } while (0)

// Q = quantized, F = float, order is src0, src1, dst
enum ggml_cuda_op_type {
    GGML_CUDA_OP_TYPE_QQQ = 0,
    GGML_CUDA_OP_TYPE_QQF = 1,
    GGML_CUDA_OP_TYPE_QFQ = 2,
    GGML_CUDA_OP_TYPE_QFF = 3,
    GGML_CUDA_OP_TYPE_FQQ = 4,
    GGML_CUDA_OP_TYPE_FQF = 5,
    GGML_CUDA_OP_TYPE_FFQ = 6,
    GGML_CUDA_OP_TYPE_FFF = 7,
};

typedef void (*dequantize_kernel_t)(const void * vx, const int ib, const int iqs, float & v0, float & v1);
typedef void (*to_fp32_cuda_t)(const void * x, float * y, int k, hipStream_t stream);
typedef void (*ggml_cuda_func_t)(const ggml_tensor * src0, const ggml_tensor * src1, ggml_tensor * dst, void * wdata, size_t wsize);
typedef void (*ggml_cuda_op_t)(
    const ggml_tensor * src0, const ggml_tensor * src1, ggml_tensor * dst, char * src0_ddq_i,
    float * src0_ddf_i, float * src1_ddf_i, float * dst_ddf_i, int i1, hipStream_t & cudaStream_main);

// QK = number of values after dequantization
// QR = QK / number of values before dequantization

#define QK4_0 32
#define QR4_0 2
typedef struct {
    half    d;              // delta
    uint8_t qs[QK4_0 / 2];  // nibbles / quants
} block_q4_0;
static_assert(sizeof(block_q4_0) == sizeof(ggml_fp16_t) + QK4_0 / 2, "wrong q4_0 block size/padding");

#define QK4_1 32
#define QR4_1 2
typedef struct {
    half    d;              // delta
    half    m;              // min
    uint8_t qs[QK4_1 / 2];  // nibbles / quants
} block_q4_1;
static_assert(sizeof(block_q4_1) == sizeof(ggml_fp16_t) * 2 + QK4_1 / 2, "wrong q4_1 block size/padding");

#define QK5_0 32
#define QR5_0 2
typedef struct {
    half d;                 // delta
    uint8_t qh[4];          // 5-th bit of quants
    uint8_t qs[QK5_0 / 2];  // nibbles / quants
} block_q5_0;
static_assert(sizeof(block_q5_0) == sizeof(ggml_fp16_t) + sizeof(uint32_t) + QK5_0 / 2, "wrong q5_0 block size/padding");

#define QK5_1 32
#define QR5_1 2
typedef struct {
    half d;                 // delta
    half m;                 // min
    uint8_t qh[4];          // 5-th bit of quants
    uint8_t qs[QK5_1 / 2];  // nibbles / quants
} block_q5_1;
static_assert(sizeof(block_q5_1) == 2 * sizeof(ggml_fp16_t) + sizeof(uint32_t) + QK5_1 / 2, "wrong q5_1 block size/padding");

#define QK8_0 32
#define QR8_0 1
typedef struct {
    half    d;              // delta
    int8_t  qs[QK8_0];      // quants
} block_q8_0;
static_assert(sizeof(block_q8_0) == sizeof(ggml_fp16_t) + QK8_0, "wrong q8_0 block size/padding");

#define WARP_SIZE 32

#define CUDA_MUL_BLOCK_SIZE 256

#define CUDA_DEQUANTIZE_BLOCK_SIZE 256

// dmmv = dequantize_mul_mat_vec
#ifndef GGML_CUDA_DMMV_X
#define GGML_CUDA_DMMV_X 32
#endif
#ifndef GGML_CUDA_DMMV_Y
#define GGML_CUDA_DMMV_Y 1
#endif

static __global__ void mul_f32(const float * x, const float * y, float * dst, const int kx, const int ky) {
    const int i = blockDim.x*blockIdx.x + threadIdx.x;

    if (i >= kx) {
        return;
    }
    dst[i] = x[i] * y[i%ky];
}

static __device__ void dequantize_q4_0(const void * vx, const int ib, const int iqs, float & v0, float & v1){
    const block_q4_0 * x = (const block_q4_0 *) vx;

    const float d = x[ib].d;

    const uint8_t vui = x[ib].qs[iqs];

    const int8_t vi0 = vui & 0xF;
    const int8_t vi1 = vui >> 4;

    v0 = (vi0 - 8)*d;
    v1 = (vi1 - 8)*d;
}

static __device__ void dequantize_q4_1(const void * vx, const int ib, const int iqs, float & v0, float & v1){
    const block_q4_1 * x = (const block_q4_1 *) vx;

    const float d = x[ib].d;
    const float m = x[ib].m;

    const uint8_t vui = x[ib].qs[iqs];

    const int8_t vi0 = vui & 0xF;
    const int8_t vi1 = vui >> 4;

    v0 = vi0*d + m;
    v1 = vi1*d + m;
}

static __device__ void dequantize_q5_0(const void * vx, const int ib, const int iqs, float & v0, float & v1){
    const block_q5_0 * x = (const block_q5_0 *) vx;

    const float d = x[ib].d;

    uint32_t qh;
    memcpy(&qh, x[ib].qh, sizeof(qh));

    const uint8_t xh_0 = ((qh >> (iqs +  0)) << 4) & 0x10;
    const uint8_t xh_1 = ((qh >> (iqs + 12))     ) & 0x10;

    const int32_t x0 = ((x[ib].qs[iqs] & 0xf) | xh_0) - 16;
    const int32_t x1 = ((x[ib].qs[iqs] >>  4) | xh_1) - 16;

    v0 = x0*d;
    v1 = x1*d;
}

static __device__ void dequantize_q5_1(const void * vx, const int ib, const int iqs, float & v0, float & v1){
    const block_q5_1 * x = (const block_q5_1 *) vx;

    const float d = x[ib].d;
    const float m = x[ib].m;

    uint32_t qh;
    memcpy(&qh, x[ib].qh, sizeof(qh));

    const uint8_t xh_0 = ((qh >> (iqs +  0)) << 4) & 0x10;
    const uint8_t xh_1 = ((qh >> (iqs + 12))     ) & 0x10;

    const int32_t x0 = ((x[ib].qs[iqs] & 0xf) | xh_0);
    const int32_t x1 = ((x[ib].qs[iqs] >>  4) | xh_1);

    v0 = x0*d + m;
    v1 = x1*d + m;
}

static __device__ void dequantize_q8_0(const void * vx, const int ib, const int iqs, float & v0, float & v1){
    const block_q8_0 * x = (const block_q8_0 *) vx;

    const float d = x[ib].d;

    const int8_t vi0 = x[ib].qs[iqs + 0];
    const int8_t vi1 = x[ib].qs[iqs + 1];

    v0 = vi0*d;
    v1 = vi1*d;
}

static __device__ void convert_f16(const void * vx, const int ib, const int iqs, float & v0, float & v1){
    const half * x = (const half *) vx;

    v0 = __half2float(x[ib + 0]);
    v1 = __half2float(x[ib + 1]);
}

template <int qk, int qr, dequantize_kernel_t dequantize_kernel>
static __global__ void dequantize_block(const void * vx, float * y, const int k) {
    const int i = blockDim.x*blockIdx.x + 2*threadIdx.x;

    if (i >= k) {
        return;
    }

    const int ib = i/qk; // block index
    const int iqs = (i%qk)/qr; // quant index
    const int iybs = i - i%qk; // y block start index
    const int y_offset = qr == 1 ? 1 : qk/2;

    // dequantize
    float & v0 = y[iybs + iqs + 0];
    float & v1 = y[iybs + iqs + y_offset];
    dequantize_kernel(vx, ib, iqs, v0, v1);
}

template <int qk, int qr, dequantize_kernel_t dequantize_kernel>
static __global__ void dequantize_mul_mat_vec(const void * vx, const float * y, float * dst, const int ncols) {
    // qk = quantized weights per x block
    // qr = number of quantized weights per data value in x block
    const int row = blockIdx.x*blockDim.y + threadIdx.y;
    const int tid = threadIdx.x;

    const int iter_stride = 2*GGML_CUDA_DMMV_X;
    const int vals_per_iter = iter_stride / WARP_SIZE; // num quantized vals per thread and i iter
    const int y_offset = qr == 1 ? 1 : qk/2;

    float tmp = 0; // partial sum for thread in warp

    for (int i = 0; i < ncols; i += iter_stride) {
        const int col = i + vals_per_iter*tid;
        const int ib = (row*ncols + col)/qk; // x block index
        const int iqs = (col%qk)/qr; // x quant index
        const int iybs = col - col%qk; // y block start index

// processing >2 values per i iter is faster for fast GPUs
#pragma unroll
        for (int j = 0; j < vals_per_iter; j += 2) {
            // process 2 vals per j iter

            // dequantize
            float v0, v1;
            dequantize_kernel(vx, ib, iqs + j/qr, v0, v1);
            // for qr = 2 the iqs needs to increase by 1 per j iter because 2 weights per data val

            // matrix multiplication
            tmp += v0 * y[iybs + iqs + j/qr + 0];
            tmp += v1 * y[iybs + iqs + j/qr + y_offset];
            // for qr = 2 the y index needs to increase by 1 per j iter because of y_offset = qk/2
        }
    }

    // sum up partial sums and write back result
    __syncthreads();
#pragma unroll
    for (int mask = 16; mask > 0; mask >>= 1) {
        tmp += __shfl_xor_sync(0xffffffff, tmp, mask, 32);
    }

    if (tid == 0) {
        dst[row] = tmp;
    }
}

static void mul_f32_cuda(const float * x, const float * y, float * dst, const int kx, const int ky, hipStream_t stream) {
    const int num_blocks = (kx + CUDA_MUL_BLOCK_SIZE - 1) / CUDA_MUL_BLOCK_SIZE;
    mul_f32<<<num_blocks, CUDA_MUL_BLOCK_SIZE, 0, stream>>>(x, y, dst, kx, ky);
}

static void dequantize_row_q4_0_cuda(const void * vx, float * y, const int k, hipStream_t stream) {
    const int num_blocks = (k + CUDA_DEQUANTIZE_BLOCK_SIZE - 1) / CUDA_DEQUANTIZE_BLOCK_SIZE;
    dequantize_block<QK4_0, QR4_0, dequantize_q4_0><<<num_blocks, CUDA_DEQUANTIZE_BLOCK_SIZE, 0, stream>>>(vx, y, k);
}

static void dequantize_row_q4_1_cuda(const void * vx, float * y, const int k, hipStream_t stream) {
    const int num_blocks = (k + CUDA_DEQUANTIZE_BLOCK_SIZE - 1) / CUDA_DEQUANTIZE_BLOCK_SIZE;
    dequantize_block<QK4_1, QR4_1, dequantize_q4_1><<<num_blocks, CUDA_DEQUANTIZE_BLOCK_SIZE, 0, stream>>>(vx, y, k);
}

static void dequantize_row_q5_0_cuda(const void * vx, float * y, const int k, hipStream_t stream) {
    const int num_blocks = (k + CUDA_DEQUANTIZE_BLOCK_SIZE - 1) / CUDA_DEQUANTIZE_BLOCK_SIZE;
    dequantize_block<QK5_0, QR5_0, dequantize_q5_0><<<num_blocks, CUDA_DEQUANTIZE_BLOCK_SIZE, 0, stream>>>(vx, y, k);
}

static void dequantize_row_q5_1_cuda(const void * vx, float * y, const int k, hipStream_t stream) {
    const int num_blocks = (k + CUDA_DEQUANTIZE_BLOCK_SIZE - 1) / CUDA_DEQUANTIZE_BLOCK_SIZE;
    dequantize_block<QK5_1, QR5_1, dequantize_q5_1><<<num_blocks, CUDA_DEQUANTIZE_BLOCK_SIZE, 0, stream>>>(vx, y, k);
}

static void dequantize_row_q8_0_cuda(const void * vx, float * y, const int k, hipStream_t stream) {
    const int num_blocks = (k + CUDA_DEQUANTIZE_BLOCK_SIZE - 1) / CUDA_DEQUANTIZE_BLOCK_SIZE;
    dequantize_block<QK8_0, QR8_0, dequantize_q8_0><<<num_blocks, CUDA_DEQUANTIZE_BLOCK_SIZE, 0, stream>>>(vx, y, k);
}

static void dequantize_mul_mat_vec_q4_0_cuda(const void * vx, const float * y, float * dst, const int ncols, const int nrows, hipStream_t stream) {
    GGML_ASSERT(ncols % GGML_CUDA_DMMV_X == 0);
    GGML_ASSERT(nrows % GGML_CUDA_DMMV_Y == 0);
    const dim3 block_dims(WARP_SIZE, GGML_CUDA_DMMV_Y, 1);
    dequantize_mul_mat_vec<QK4_0, QR4_0, dequantize_q4_0>
        <<<nrows/GGML_CUDA_DMMV_Y, block_dims, 0, stream>>>(vx, y, dst, ncols);
}

static void dequantize_mul_mat_vec_q4_1_cuda(const void * vx, const float * y, float * dst, const int ncols, const int nrows, hipStream_t stream) {
    GGML_ASSERT(ncols % GGML_CUDA_DMMV_X == 0);
    GGML_ASSERT(nrows % GGML_CUDA_DMMV_Y == 0);
    const dim3 block_dims(WARP_SIZE, GGML_CUDA_DMMV_Y, 1);
    dequantize_mul_mat_vec<QK4_1, QR4_1, dequantize_q4_1>
        <<<nrows/GGML_CUDA_DMMV_Y, block_dims, 0, stream>>>(vx, y, dst, ncols);
}

static void dequantize_mul_mat_vec_q5_0_cuda(const void * vx, const float * y, float * dst, const int ncols, const int nrows, hipStream_t stream) {
    GGML_ASSERT(ncols % GGML_CUDA_DMMV_X == 0);
    GGML_ASSERT(nrows % GGML_CUDA_DMMV_Y == 0);
    const dim3 block_dims(WARP_SIZE, GGML_CUDA_DMMV_Y, 1);
    dequantize_mul_mat_vec<QK5_0, QR5_0, dequantize_q5_0>
        <<<nrows/GGML_CUDA_DMMV_Y, block_dims, 0, stream>>>(vx, y, dst, ncols);
}

static void dequantize_mul_mat_vec_q5_1_cuda(const void * vx, const float * y, float * dst, const int ncols, const int nrows, hipStream_t stream) {
    GGML_ASSERT(ncols % GGML_CUDA_DMMV_X == 0);
    GGML_ASSERT(nrows % GGML_CUDA_DMMV_Y == 0);
    const dim3 block_dims(WARP_SIZE, GGML_CUDA_DMMV_Y, 1);
    dequantize_mul_mat_vec<QK5_1, QR5_1, dequantize_q5_1>
        <<<nrows/GGML_CUDA_DMMV_Y, block_dims, 0, stream>>>(vx, y, dst, ncols);
}

static void dequantize_mul_mat_vec_q8_0_cuda(const void * vx, const float * y, float * dst, const int ncols, const int nrows, hipStream_t stream) {
    GGML_ASSERT(ncols % GGML_CUDA_DMMV_X == 0);
    GGML_ASSERT(nrows % GGML_CUDA_DMMV_Y == 0);
    const dim3 block_dims(WARP_SIZE, GGML_CUDA_DMMV_Y, 1);
    dequantize_mul_mat_vec<QK8_0, QR8_0, dequantize_q8_0>
        <<<nrows/GGML_CUDA_DMMV_Y, block_dims, 0, stream>>>(vx, y, dst, ncols);
}

static void convert_fp16_to_fp32_cuda(const void * vx, float * y, const int k, hipStream_t stream) {
    const int num_blocks = (k + CUDA_DEQUANTIZE_BLOCK_SIZE - 1) / CUDA_DEQUANTIZE_BLOCK_SIZE;
    dequantize_block<32, 1, convert_f16><<<num_blocks, CUDA_DEQUANTIZE_BLOCK_SIZE, 0, stream>>>(vx, y, k);
}

static void convert_mul_mat_vec_f16_cuda(const void * vx, const float * y, float * dst, const int ncols, const int nrows, hipStream_t stream) {
    GGML_ASSERT(ncols % GGML_CUDA_DMMV_X == 0);
    GGML_ASSERT(nrows % GGML_CUDA_DMMV_Y == 0);
    const dim3 block_dims(WARP_SIZE, GGML_CUDA_DMMV_Y, 1);
    dequantize_mul_mat_vec<1, 1, convert_f16>
        <<<nrows/GGML_CUDA_DMMV_Y, block_dims, 0, stream>>>(vx, y, dst, ncols);
}

static to_fp32_cuda_t ggml_get_to_fp32_cuda(ggml_type type) {
    switch (type) {
        case GGML_TYPE_Q4_0:
            return dequantize_row_q4_0_cuda;
        case GGML_TYPE_Q4_1:
            return dequantize_row_q4_1_cuda;
        case GGML_TYPE_Q5_0:
            return dequantize_row_q5_0_cuda;
        case GGML_TYPE_Q5_1:
            return dequantize_row_q5_1_cuda;
        case GGML_TYPE_Q8_0:
            return dequantize_row_q8_0_cuda;
        case GGML_TYPE_F16:
            return convert_fp16_to_fp32_cuda;
        default:
            return nullptr;
    }
}

// buffer pool for cuda
#define MAX_CUDA_BUFFERS 256

struct scoped_spin_lock {
    std::atomic_flag& lock;
    scoped_spin_lock(std::atomic_flag& lock) : lock(lock) {
        while (lock.test_and_set(std::memory_order_acquire)) {
            ; // spin
        }
    }
    ~scoped_spin_lock() {
        lock.clear(std::memory_order_release);
    }
    scoped_spin_lock(const scoped_spin_lock&) = delete;
    scoped_spin_lock& operator=(const scoped_spin_lock&) = delete;
};

struct cuda_buffer {
    void * ptr = nullptr;
    size_t size = 0;
};

static cuda_buffer g_cuda_buffer_pool[MAX_CUDA_BUFFERS];
static std::atomic_flag g_cuda_pool_lock = ATOMIC_FLAG_INIT;

static void * ggml_cuda_pool_malloc(size_t size, size_t * actual_size) {
    scoped_spin_lock lock(g_cuda_pool_lock);

    for (int i = 0; i < MAX_CUDA_BUFFERS; ++i) {
        cuda_buffer& b = g_cuda_buffer_pool[i];
        if (b.size >= size && b.ptr != nullptr) {
            void * ptr = b.ptr;
            *actual_size = b.size;
            b.ptr = nullptr;
            b.size = 0;
            return ptr;
        }
    }
    void * ptr;
    CUDA_CHECK(hipMalloc((void **) &ptr, size));
    *actual_size = size;
    return ptr;
}

static void ggml_cuda_pool_free(void * ptr, size_t size) {
    scoped_spin_lock lock(g_cuda_pool_lock);

    for (int i = 0; i < MAX_CUDA_BUFFERS; ++i) {
        cuda_buffer& b = g_cuda_buffer_pool[i];
        if (b.ptr == nullptr) {
            b.ptr = ptr;
            b.size = size;
            return;
        }
    }
    fprintf(stderr, "WARNING: cuda buffer pool full, increase MAX_CUDA_BUFFERS\n");
    CUDA_CHECK(hipFree(ptr));
}

#define GGML_CUDA_MAX_STREAMS 8 // Set this to 1 for reproducible matrix multiplication.
#define GGML_CUDA_MAX_EVENTS 64

static int g_device_count = -1;
static float g_vram_splits[GGML_MAX_DEVICES];

static hipblasHandle_t g_cublasH = nullptr;

static hipStream_t g_cudaStreams_main[GGML_CUDA_MAX_STREAMS] = { nullptr };
static hipEvent_t g_cudaEvents_main[GGML_CUDA_MAX_EVENTS] = { nullptr };

static hipStream_t g_cudaStreams_memcpy_src1[GGML_CUDA_MAX_STREAMS] = { nullptr };
static hipEvent_t g_cudaEvents_memcpy_src1[GGML_CUDA_MAX_EVENTS] = { nullptr };

static hipStream_t g_cudaStreams_memcpy_dst[GGML_CUDA_MAX_STREAMS] = { nullptr };

void ggml_init_cublas() {
    static bool initialized = false;

    if (!initialized) {
        CUDA_CHECK(hipGetDeviceCount(&g_device_count));
        int64_t total_vram = 0;
        fprintf(stderr, "%s: found %d CUDA devices:\n", __func__, g_device_count);
        for (int i = 0; i < g_device_count; ++i) {
            hipDeviceProp_t prop;
            CUDA_CHECK(hipGetDeviceProperties(&prop, i));
            fprintf(stderr, "  %d. %s\n", i+1, prop.name);
            g_vram_splits[i] = total_vram;
            total_vram += prop.totalGlobalMem;
        }
        for (int i = 0; i < g_device_count; ++i) {
            g_vram_splits[i] /= total_vram;
        }

        // create streams
        for (int i = 0; i < GGML_CUDA_MAX_STREAMS; ++i) {
            CUDA_CHECK(hipStreamCreateWithFlags(&g_cudaStreams_main[i], hipStreamNonBlocking));
            CUDA_CHECK(hipStreamCreateWithFlags(&g_cudaStreams_memcpy_src1[i], hipStreamNonBlocking));
            CUDA_CHECK(hipStreamCreateWithFlags(&g_cudaStreams_memcpy_dst[i], hipStreamNonBlocking));
        }
        // create events
        for (int i = 0; i < GGML_CUDA_MAX_EVENTS; ++i) {
            CUDA_CHECK(hipEventCreateWithFlags(&g_cudaEvents_main[i], hipEventDisableTiming));
            CUDA_CHECK(hipEventCreateWithFlags(&g_cudaEvents_memcpy_src1[i], hipEventDisableTiming));
        }

        // create cublas handle
        CUBLAS_CHECK(hipblasCreate(&g_cublasH));
        CUBLAS_CHECK(hipblasSetMathMode(g_cublasH, HIPBLAS_TF32_TENSOR_OP_MATH));

        // configure logging to stdout
        // CUBLAS_CHECK(cublasLoggerConfigure(1, 1, 0, nullptr));

        initialized = true;
    }
}

void * ggml_cuda_host_malloc(size_t size) {
    if (getenv("GGML_CUDA_NO_PINNED") != nullptr) {
        return nullptr;
    }

    void * ptr = nullptr;
    hipError_t err = hipHostMalloc((void **) &ptr, size);
    if (err != hipSuccess) {
        fprintf(stderr, "WARNING: failed to allocate %.2f MB of pinned memory: %s\n",
            size/1024.0/1024.0, hipGetErrorString(err));
        return nullptr;
    }

    return ptr;
}

void ggml_cuda_host_free(void * ptr) {
    CUDA_CHECK(hipHostFree(ptr));
}

static hipError_t ggml_cuda_h2d_tensor_2d(void * dst, const struct ggml_tensor * src, uint64_t i3, uint64_t i2, hipStream_t stream) {
    const uint64_t ne0 = src->ne[0];
    const uint64_t ne1 = src->ne[1];
    const uint64_t nb0 = src->nb[0];
    const uint64_t nb1 = src->nb[1];
    const uint64_t nb2 = src->nb[2];
    const uint64_t nb3 = src->nb[3];
    const enum ggml_type type = src->type;
    const size_t ts = ggml_type_size(type);
    const size_t bs = ggml_blck_size(type);

    const void * x = (const void *) ((const char *) src->data + i2*nb2 + i3*nb3);
    if (nb0 == ts && nb1 == ts*ne0/bs) {
        return hipMemcpyAsync(dst, x, ne1*nb1, hipMemcpyHostToDevice, stream);
    } else if (nb0 == ts) {
        return hipMemcpy2DAsync(dst, ts*ne0/bs, x, nb1, ts*ne0/bs, ne1, hipMemcpyHostToDevice, stream);
    } else {
        for (uint64_t i1 = 0; i1 < ne1; i1++) {
            const void * rx = (const void *) ((const char *) x + i1*nb1);
            void * rd = (void *) ((char *) dst + i1*ts*ne0/bs);
            // pretend the row is a matrix with cols=1
            hipError_t r = hipMemcpy2DAsync(rd, ts/bs, rx, nb0, ts/bs, ne0, hipMemcpyHostToDevice, stream);
            if (r != hipSuccess) return r;
        }
        return hipSuccess;
    }
}

static void ggml_cuda_mul_mat_f16(const ggml_tensor * src0, const ggml_tensor * src1, ggml_tensor * dst, void * wdata, size_t /* wsize */) {
    const int64_t ne00 = src0->ne[0];
    const int64_t ne01 = src0->ne[1];
    const int64_t ne02 = src0->ne[2];
    const int64_t ne03 = src0->ne[3];

    const int64_t ne10 = src1->ne[0];
    const int64_t ne11 = src1->ne[1];

    const int nb10 = src1->nb[0];
    const int nb11 = src1->nb[1];
    const int nb12 = src1->nb[2];
    const int nb13 = src1->nb[3];

    const int nb2  = dst->nb[2];
    const int nb3  = dst->nb[3];

    const float alpha = 1.0f;
    const float beta = 0.0f;
    const int x_ne = ne01 * ne00;
    const int y_ne = ne11 * ne10;
    const int d_ne = ne11 * ne01;
    const int n_mm = ne03 * ne02;

    size_t x_size, y_size, d_size;
    half  * d_X =  (half *) ggml_cuda_pool_malloc(n_mm * sizeof(half) * x_ne, &x_size);
    half  * d_Y =  (half *) ggml_cuda_pool_malloc(n_mm * sizeof(half) * y_ne, &y_size);
    float * d_D = (float *) ggml_cuda_pool_malloc(n_mm * sizeof(float) * d_ne, &d_size);

    bool src1_cont_rows = nb10 == sizeof(float);
    bool src1_cont_cols = (size_t)nb11 == ne11*sizeof(float);

    for (int64_t i03 = 0; i03 < ne03; i03++) {
        for (int64_t i02 = 0; i02 < ne02; i02++) {
            int i = i03*ne02 + i02;
            hipStream_t cudaStream = g_cudaStreams_main[i % GGML_CUDA_MAX_STREAMS];

            half  * c_X = d_X + i * x_ne;
            half  * c_Y = d_Y + i * y_ne;
            float * c_D = d_D + i * d_ne;

            // copy src0 to device
            CUDA_CHECK(ggml_cuda_h2d_tensor_2d(c_X, src0, i03, i02, cudaStream));

            // convert src1 to fp16
            // TODO: use multiple threads
            ggml_fp16_t * const tmp = (ggml_fp16_t *) wdata + (ne11 * ne10) * (i03 * ne02 + i02);
            char * src1i = (char *) src1->data + i03*nb13 + i02*nb12;
            if (src1_cont_rows) {
                if (src1_cont_cols) {
                    ggml_fp32_to_fp16_row((float *) src1i, tmp, ne10*ne11);
                }
                else {
                    for (int64_t i01 = 0; i01 < ne11; i01++) {
                        ggml_fp32_to_fp16_row((float *) (src1i + i01*nb11), tmp + i01*ne10, ne10);
                    }
                }
            }
            else {
                for (int64_t i01 = 0; i01 < ne11; i01++) {
                    for (int64_t i00 = 0; i00 < ne10; i00++) {
                        // very slow due to no inlining
                        tmp[i01*ne10 + i00] = ggml_fp32_to_fp16(*(float *) (src1i + i01*nb11 + i00*nb10));
                    }
                }
            }

            // copy src1 to device
            CUDA_CHECK(hipMemcpyAsync(c_Y, tmp, sizeof(half) * y_ne, hipMemcpyHostToDevice, cudaStream));

            // compute
            CUBLAS_CHECK(hipblasSetStream(g_cublasH, cudaStream));
            CUBLAS_CHECK(
                hipblasGemmEx(g_cublasH, HIPBLAS_OP_T, HIPBLAS_OP_N,
                        ne01, ne11, ne10,
                        &alpha, c_X, HIP_R_16F, ne00,
                                c_Y, HIP_R_16F, ne10,
                        &beta,  c_D, HIP_R_32F, ne01,
                        HIPBLAS_COMPUTE_32F_FAST_16F,
                        HIPBLAS_GEMM_DEFAULT));

            // copy dst to host
            float * d = (float *) ((char *) dst->data + i02*nb2 + i03*nb3);
            CUDA_CHECK(hipMemcpyAsync(d, c_D, sizeof(float) * d_ne, hipMemcpyDeviceToHost, cudaStream));
        }
    }

    CUDA_CHECK(hipDeviceSynchronize());
    ggml_cuda_pool_free(d_X, x_size);
    ggml_cuda_pool_free(d_Y, y_size);
    ggml_cuda_pool_free(d_D, d_size);
}

inline void ggml_cuda_op_mul(
    const ggml_tensor * src0, const ggml_tensor * src1, ggml_tensor * dst, char * src0_ddq_i,
    float * src0_ddf_i, float * src1_ddf_i, float * dst_ddf_i, int i1, hipStream_t & cudaStream_main){

    GGML_ASSERT(src0_ddf_i != nullptr);
    GGML_ASSERT(src1_ddf_i != nullptr);
    GGML_ASSERT(dst_ddf_i != nullptr);

    const int64_t ne00 = src0->ne[0];
    const int64_t ne01 = src0->ne[1];

    const int64_t ne10 = src1->ne[0];
    const int64_t ne11 = src1->ne[1];

    for (int64_t i01 = 0; i01 < ne01; i01++) {
        const int64_t i11 = i1*ne11 + i01%ne11; // broadcast src1 across src0

        float * src0_ddf_i01 = src0_ddf_i + i01*ne00;
        float * src1_ddf_i01 = src1_ddf_i + i11*ne10;
        float * dst_ddf_i01 = dst_ddf_i + i01*ne00;

        // compute
        mul_f32_cuda(src0_ddf_i01, src1_ddf_i01, dst_ddf_i01, ne00, ne10, cudaStream_main);
        CUDA_CHECK(hipGetLastError());
    }

    (void) dst;
    (void) src0_ddq_i;
}

inline void ggml_cuda_op_dequantize_mul_mat_vec(
    const ggml_tensor * src0, const ggml_tensor * src1, ggml_tensor * dst, char * src0_ddq_i,
    float * src0_ddf_i, float * src1_ddf_i, float * dst_ddf_i, int i1, hipStream_t & cudaStream_main){

    GGML_ASSERT(src0_ddq_i != nullptr);
    GGML_ASSERT(src1_ddf_i != nullptr);
    GGML_ASSERT(dst_ddf_i != nullptr);

    const int64_t ne00 = src0->ne[0];
    const int64_t ne01 = src0->ne[1];

    switch (src0->type) {
        case GGML_TYPE_Q4_0:
            dequantize_mul_mat_vec_q4_0_cuda(src0_ddq_i, src1_ddf_i, dst_ddf_i, ne00, ne01, cudaStream_main);
            break;
        case GGML_TYPE_Q4_1:
            dequantize_mul_mat_vec_q4_1_cuda(src0_ddq_i, src1_ddf_i, dst_ddf_i, ne00, ne01, cudaStream_main);
            break;
        case GGML_TYPE_Q5_0:
            dequantize_mul_mat_vec_q5_0_cuda(src0_ddq_i, src1_ddf_i, dst_ddf_i, ne00, ne01, cudaStream_main);
            break;
        case GGML_TYPE_Q5_1:
            dequantize_mul_mat_vec_q5_1_cuda(src0_ddq_i, src1_ddf_i, dst_ddf_i, ne00, ne01, cudaStream_main);
            break;
        case GGML_TYPE_Q8_0:
            dequantize_mul_mat_vec_q8_0_cuda(src0_ddq_i, src1_ddf_i, dst_ddf_i, ne00, ne01, cudaStream_main);
            break;
        case GGML_TYPE_F16:
            convert_mul_mat_vec_f16_cuda(src0_ddq_i, src1_ddf_i, dst_ddf_i, ne00, ne01, cudaStream_main);
            break;
        default:
            GGML_ASSERT(false);
            break;
    }
    CUDA_CHECK(hipGetLastError());

    (void) src1;
    (void) dst;
    (void) src0_ddf_i;
    (void) i1;
}

inline void ggml_cuda_op_mul_mat_cublas(
    const ggml_tensor * src0, const ggml_tensor * src1, ggml_tensor * dst, char * src0_ddq_i,
    float * src0_ddf_i, float * src1_ddf_i, float * dst_ddf_i, int i1, hipStream_t & cudaStream_main){

    GGML_ASSERT(src0_ddf_i != nullptr);
    GGML_ASSERT(src1_ddf_i != nullptr);
    GGML_ASSERT(dst_ddf_i != nullptr);

    const float alpha = 1.0f;
    const float beta = 0.0f;

    const int64_t ne00 = src0->ne[0];
    const int64_t ne01 = src0->ne[1];

    const int64_t ne10 = src1->ne[0];
    const int64_t ne11 = src1->ne[1];

    CUBLAS_CHECK(hipblasSetStream(g_cublasH, cudaStream_main));
    CUBLAS_CHECK(
        hipblasSgemm(g_cublasH, HIPBLAS_OP_T, HIPBLAS_OP_N,
                ne01, ne11, ne10,
                &alpha, src0_ddf_i, ne00,
                        src1_ddf_i, ne10,
                &beta,  dst_ddf_i,  ne01));

    (void) dst;
    (void) src0_ddq_i;
    (void) i1;
}

template<enum ggml_cuda_op_type op_type, ggml_cuda_op_t op>
static void ggml_cuda_op(const ggml_tensor * src0, const ggml_tensor * src1, ggml_tensor * dst) {
    const int64_t ne00 = src0->ne[0];
    const int64_t ne01 = src0->ne[1];
    const int64_t ne02 = src0->ne[2];
    const int64_t ne03 = src0->ne[3];

    const int64_t ne10 = src1->ne[0];
    const int64_t ne11 = src1->ne[1];
    const int64_t ne12 = src1->ne[2];
    const int64_t ne13 = src1->ne[3];

    const int64_t ne0 = dst->ne[0];
    const int64_t ne1 = dst->ne[1];

    const int nb2  = dst->nb[2];
    const int nb3  = dst->nb[3];

    const int64_t src0_stride = ne00 * ne01;
    const int64_t src1_stride = ne10 * ne11;
    const int64_t dst_stride = ne0 * ne1;
    const int64_t num_iters = ne02 * ne03;

    const size_t src0_ts = ggml_type_size(src0->type);

    const bool src0_on_device = src0->backend == GGML_BACKEND_CUDA;
    const bool src0_is_f32 = src0->type == GGML_TYPE_F32;
    const bool src0_needs_f32 = op_type & 0x4; // 3rd least significant bit = src0 needs f32

    const bool src1_on_device = src1->backend == GGML_BACKEND_CUDA;

    const bool dst_on_device = dst->backend == GGML_BACKEND_CUDA;

    const to_fp32_cuda_t to_fp32_cuda = ggml_get_to_fp32_cuda(src0->type);

    // dd = data device
    char  * src0_ddq = nullptr; // quantized
    float * src0_ddf = nullptr; // float
    float * src1_ddf = nullptr;
    float * dst_ddf = nullptr;

    bool src0_ddq_malloced = false;
    bool src0_ddf_malloced = false;
    bool src1_ddf_malloced = false;
    bool dst_ddf_malloced = false;

    // asq = actual size quantized, asf = actual size float
    size_t src0_asq, src0_asf, src1_asf, dst_asf;

    if (src0_on_device) {
        if (src0_is_f32) {
            src0_ddf = (float *) src0->data_device[0];
        } else {
            src0_ddq = (char *) src0->data_device[0];
        }
    } else {
        if (src0_is_f32) {
            src0_ddf = (float *) ggml_cuda_pool_malloc(num_iters * src0_stride * sizeof(float), &src0_asf);
            src0_ddf_malloced = true;
        } else {
            src0_ddq = (char *) ggml_cuda_pool_malloc(num_iters * src0_stride * src0_ts, &src0_asq);
            src0_ddq_malloced = true;
        }
    }

    if (src0_needs_f32 && !src0_is_f32) {
        src0_ddf = (float *) ggml_cuda_pool_malloc(num_iters * src0_stride * sizeof(float), &src0_asf);
        src0_ddf_malloced = true;
    }

    if (src1_on_device) {
        src1_ddf = (float *) src1->data_device[0];
    } else {
        src1_ddf = (float *) ggml_cuda_pool_malloc(num_iters * src1_stride * sizeof(float), &src1_asf);
        src1_ddf_malloced = true;
    }
    if (dst_on_device) {
        dst_ddf = (float *) dst->data_device[0];
    } else {
        dst_ddf = (float *) ggml_cuda_pool_malloc(num_iters * dst_stride * sizeof(float), &dst_asf);
        dst_ddf_malloced = true;
    }

    for (int64_t i03 = 0; i03 < ne03; i03++) {
        const int64_t i13 = i03 % ne13;
        for (int64_t i02 = 0; i02 < ne02; i02++) {
            const int64_t i12 = i02 % ne12;

            const int64_t i0 = i03*ne02 + i02;
            const int64_t i1 = i13*ne12 + i12;

            hipStream_t cudaStream_main = g_cudaStreams_main[i0 % GGML_CUDA_MAX_STREAMS];
            hipStream_t cudaStream_memcpy_src1 = g_cudaStreams_memcpy_src1[i0 % GGML_CUDA_MAX_STREAMS];
            hipStream_t cudaStream_memcpy_dst = g_cudaStreams_memcpy_dst[i0 % GGML_CUDA_MAX_STREAMS];
            hipEvent_t  cudaEvent_main = g_cudaEvents_main[i0 % GGML_CUDA_MAX_EVENTS];
            hipEvent_t  cudaEvent_memcpy_src1 = g_cudaEvents_memcpy_src1[i0 % GGML_CUDA_MAX_EVENTS];

            char  * src0_ddq_i = src0_ddq + i0*src0_stride;
            float * src0_ddf_i = src0_ddf + i0*src0_stride;
            float * src1_ddf_i = src1_ddf + i1*src1_stride;
            float * dst_ddf_i = dst_ddf + i0*dst_stride;

            // copy src0, src1 to device if necessary
            if (!src1_on_device) { // src1 first to avoid blocking device queues
                    CUDA_CHECK(ggml_cuda_h2d_tensor_2d(src1_ddf, src1, i03, i02, cudaStream_memcpy_src1));
            }
            CUDA_CHECK(hipEventRecord(cudaEvent_memcpy_src1, cudaStream_memcpy_src1));
            if (!src0_on_device) {
                if (src0_is_f32) {
                    CUDA_CHECK(ggml_cuda_h2d_tensor_2d(src0_ddf, src0, i03, i02, cudaStream_main));
                } else {
                    CUDA_CHECK(ggml_cuda_h2d_tensor_2d(src0_ddq, src0, i03, i02, cudaStream_main));
                }
            }

            if (src0_needs_f32 && !src0_is_f32) {
                to_fp32_cuda(src0_ddq_i, src0_ddf_i, src0_stride, cudaStream_main);
                CUDA_CHECK(hipGetLastError());
            }

            // wait with main stream until src1 memcpy is done
            CUDA_CHECK(hipStreamWaitEvent(cudaStream_main, cudaEvent_memcpy_src1, 0));

            // do the computation
            op(src0, src1, dst, src0_ddq_i, src0_ddf_i, src1_ddf_i, dst_ddf_i, i1, cudaStream_main);

            CUDA_CHECK(hipEventRecord(cudaEvent_main, cudaStream_main));

            // copy dst to host if necessary
            if (!dst_on_device) {
                // wait with memcpy until main stream is done
                CUDA_CHECK(hipStreamWaitEvent(cudaStream_memcpy_dst, cudaEvent_main, 0));

                float * dhf_dst_i = (float *) ((char *) dst->data + i02*nb2 + i03*nb3);
                CUDA_CHECK(hipMemcpyAsync(dhf_dst_i, dst_ddf_i, dst_stride*sizeof(float), hipMemcpyDeviceToHost, cudaStream_memcpy_dst));
            }
        }
    }

    CUDA_CHECK(hipDeviceSynchronize());
    if (src0_ddf_malloced) {
        ggml_cuda_pool_free(src0_ddf, src0_asf);
    }
    if (src0_ddq_malloced) {
        ggml_cuda_pool_free(src0_ddq, src0_asq);
    }
    if (src1_ddf_malloced) {
        ggml_cuda_pool_free(src1_ddf, src1_asf);
    }
    if (dst_ddf_malloced) {
        ggml_cuda_pool_free(dst_ddf, dst_asf);
    }
}

bool ggml_cuda_can_mul(const struct ggml_tensor * src0, const struct ggml_tensor * src1, struct ggml_tensor * dst) {
    (void) src0;
    (void) dst;
    return src1->backend == GGML_BACKEND_CUDA;
}

void ggml_cuda_mul(const ggml_tensor * src0, const ggml_tensor * src1, ggml_tensor * dst, void * wdata, size_t wsize) {
    GGML_ASSERT(src0->type == GGML_TYPE_F32 && src1->type == GGML_TYPE_F32 && dst->type == GGML_TYPE_F32);
    ggml_cuda_op<GGML_CUDA_OP_TYPE_FFF, ggml_cuda_op_mul>(src0, src1, dst);
    (void) wdata;
    (void) wsize;
}

bool ggml_cuda_can_mul_mat(const struct ggml_tensor * src0, const struct ggml_tensor * src1, struct ggml_tensor * dst) {
    const int64_t ne10 = src1->ne[0];

    const int64_t ne0 = dst->ne[0];
    const int64_t ne1 = dst->ne[1];

    // TODO: find the optimal values for these
    if ((src0->type == GGML_TYPE_F32 || src0->type == GGML_TYPE_F16 || ggml_is_quantized(src0->type)) &&
        src1->type == GGML_TYPE_F32 &&
        dst->type == GGML_TYPE_F32 &&
        ((ne0 >= 32 && ne1 >= 32 && ne10 >= 32) || src0->backend == GGML_BACKEND_CUDA)) {
        return true;
    }

    return false;
}

bool ggml_cuda_mul_mat_use_f16(const struct ggml_tensor * src0, const struct ggml_tensor * src1, struct ggml_tensor * /* dst */) {
    size_t src0_sz = ggml_nbytes(src0);
    size_t src1_sz = ggml_nbytes(src1);

    // mul_mat_q: src0 is converted to fp32 on device
    size_t mul_mat_q_transfer = src0_sz + src1_sz;

    // mul_mat_f16: src1 is converted to fp16 on cpu
    size_t mul_mat_f16_transfer = src0_sz + sizeof(half) * ggml_nelements(src1);

    // choose the smaller one to transfer to the device
    // TODO: this is not always the best choice due to the overhead of converting to fp16
    return mul_mat_f16_transfer < mul_mat_q_transfer;
}

void ggml_cuda_mul_mat(const ggml_tensor * src0, const ggml_tensor * src1, ggml_tensor * dst, void * wdata, size_t wsize) {
    GGML_ASSERT(ggml_cuda_can_mul_mat(src0, src1, dst));

    if (src0->type == GGML_TYPE_F32) {
        ggml_cuda_op<GGML_CUDA_OP_TYPE_FFF, ggml_cuda_op_mul_mat_cublas>(src0, src1, dst);
    }
    else if (src0->type == GGML_TYPE_F16) {
        if (ggml_cuda_mul_mat_use_f16(src0, src1, dst)) {
            // ggml_cuda_op<GGML_CUDA_OP_TYPE_QQF, ggml_cuda_op_mul_mat_cublas>(src0, src1, dst);
            ggml_cuda_mul_mat_f16(src0, src1, dst, wdata, wsize);
        }
        else {
            if (src1->ne[1] == 1) {
                ggml_cuda_op<GGML_CUDA_OP_TYPE_QFF, ggml_cuda_op_dequantize_mul_mat_vec>(src0, src1, dst);
            } else {
                ggml_cuda_op<GGML_CUDA_OP_TYPE_FFF, ggml_cuda_op_mul_mat_cublas>(src0, src1, dst);
            }
        }
    }
    else if (ggml_is_quantized(src0->type)) {
        if (src1->ne[1] == 1) {
            ggml_cuda_op<GGML_CUDA_OP_TYPE_QFF, ggml_cuda_op_dequantize_mul_mat_vec>(src0, src1, dst);
        } else {
            ggml_cuda_op<GGML_CUDA_OP_TYPE_FFF, ggml_cuda_op_mul_mat_cublas>(src0, src1, dst);
        }
    }
    else {
        GGML_ASSERT(false);
    }
}

size_t ggml_cuda_mul_mat_get_wsize(const struct ggml_tensor * src0, const struct ggml_tensor * src1, struct ggml_tensor * dst) {
    if (ggml_cuda_mul_mat_use_f16(src0, src1, dst)) {
        return ggml_nelements(src1) * sizeof(ggml_fp16_t);
    }
    else {
        return 0;
    }
}

void ggml_cuda_load_data(const char * fname, struct ggml_tensor * tensor, const size_t offset, int n_layer) {
    int id = 0;
    while (id < g_device_count - 1) {
        int max_layer = n_layer*g_vram_splits[id + 1];
        if (max_layer < tensor->layer) {
            break;
        }

        ++id;
    }
    hipSetDevice(id);

    FILE * fp = fopen(fname, "rb");

    const size_t size = ggml_nbytes(tensor);

    void * buf;
    CUDA_CHECK(hipMalloc(&buf, size));
    void * buf_host = malloc(size);

#ifdef _WIN32
    int ret = _fseeki64(fp, (__int64) offset, SEEK_SET);
#else
    int ret = fseek(fp, (long) offset, SEEK_SET);
#endif
    GGML_ASSERT(ret == 0); // same

    size_t ret2 = fread(buf_host, size, 1, fp);
    if (ret2 != 1) {
        fprintf(stderr, "unexpectedly reached end of file");
        exit(1);
    }

    hipMemcpy(buf, buf_host, size, hipMemcpyHostToDevice);
    hipDeviceSynchronize();

    // tensor->data = buf;
    // free(buf_host);
    tensor->data = buf_host;
    tensor->data_device[0] = buf;
    fclose(fp);
}

bool ggml_cuda_compute_forward(struct ggml_compute_params * params, struct ggml_tensor * tensor){
    ggml_cuda_func_t func;

    switch (tensor->op) {
        case GGML_OP_MUL:
            if (!ggml_cuda_can_mul(tensor->src0, tensor->src1, tensor)) {
                return false;
            }
            func = ggml_cuda_mul;
            break;
        case GGML_OP_MUL_MAT:
            if (!ggml_cuda_can_mul_mat(tensor->src0, tensor->src1, tensor)) {
                return false;
            }
            func = ggml_cuda_mul_mat;
            break;
        default:
            return false;
    }

    if (params->ith != 0) {
        return true;
    }
    if (params->type == GGML_TASK_INIT || params->type == GGML_TASK_FINALIZE) {
        return true;
    }
    func(tensor->src0, tensor->src1, tensor, params->wdata, params->wsize);
    return true;
}
