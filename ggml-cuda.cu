#include "hip/hip_runtime.h"
#include <cstddef>
#include <cstdint>
#include <stdint.h>
#include <stdio.h>
#include <atomic>

#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hip/hip_fp16.h>

#include "ggml-cuda.h"
#include "ggml.h"

static_assert(sizeof(half) == sizeof(ggml_fp16_t), "wrong fp16 size");

#define CUDA_CHECK(err)                                                                 \
    do {                                                                                \
        hipError_t err_ = (err);                                                       \
        if (err_ != hipSuccess) {                                                      \
            fprintf(stderr, "CUDA error %d at %s:%d: %s\n", err_, __FILE__, __LINE__,   \
                hipGetErrorString(err_));                                              \
            exit(1);                                                                    \
        }                                                                               \
    } while (0)

#if CUDART_VERSION >= 12
#define CUBLAS_CHECK(err)                                                               \
    do {                                                                                \
        hipblasStatus_t err_ = (err);                                                    \
        if (err_ != HIPBLAS_STATUS_SUCCESS) {                                            \
            fprintf(stderr, "\ncuBLAS error %d at %s:%d: %s\n",                         \
                    err_, __FILE__, __LINE__, cublasGetStatusString(err_));             \
            exit(1);                                                                    \
        }                                                                               \
    } while (0)
#else
#define CUBLAS_CHECK(err)                                                               \
    do {                                                                                \
        hipblasStatus_t err_ = (err);                                                    \
        if (err_ != HIPBLAS_STATUS_SUCCESS) {                                            \
            fprintf(stderr, "\ncuBLAS error %d at %s:%d\n", err_, __FILE__, __LINE__);  \
            exit(1);                                                                    \
        }                                                                               \
    } while (0)
#endif // CUDART_VERSION >= 11

typedef void (*dequantize_kernel_t)(const void * vx, const int ib, const int iqs, float & v0, float & v1);
typedef void (*to_fp32_cuda_t)(const void * x, float * y, int k, hipStream_t stream);
typedef void (*ggml_cuda_func_t)(const ggml_tensor * src0, const ggml_tensor * src1, ggml_tensor * dst);
typedef void (*ggml_cuda_op_t)(
    const ggml_tensor * src0, const ggml_tensor * src1, ggml_tensor * dst, char * src0_ddq_i,
    float * src0_ddf_i, float * src1_ddf_i, float * dst_ddf_i, int64_t i0_low, int64_t i0_high, int i1, hipStream_t & cudaStream_main);

// QK = number of values after dequantization
// QR = QK / number of values before dequantization

#define QK4_0 32
#define QR4_0 2
typedef struct {
    half    d;              // delta
    uint8_t qs[QK4_0 / 2];  // nibbles / quants
} block_q4_0;
static_assert(sizeof(block_q4_0) == sizeof(ggml_fp16_t) + QK4_0 / 2, "wrong q4_0 block size/padding");

#define QK4_1 32
#define QR4_1 2
typedef struct {
    half    d;              // delta
    half    m;              // min
    uint8_t qs[QK4_1 / 2];  // nibbles / quants
} block_q4_1;
static_assert(sizeof(block_q4_1) == sizeof(ggml_fp16_t) * 2 + QK4_1 / 2, "wrong q4_1 block size/padding");

#define QK5_0 32
#define QR5_0 2
typedef struct {
    half d;                 // delta
    uint8_t qh[4];          // 5-th bit of quants
    uint8_t qs[QK5_0 / 2];  // nibbles / quants
} block_q5_0;
static_assert(sizeof(block_q5_0) == sizeof(ggml_fp16_t) + sizeof(uint32_t) + QK5_0 / 2, "wrong q5_0 block size/padding");

#define QK5_1 32
#define QR5_1 2
typedef struct {
    half d;                 // delta
    half m;                 // min
    uint8_t qh[4];          // 5-th bit of quants
    uint8_t qs[QK5_1 / 2];  // nibbles / quants
} block_q5_1;
static_assert(sizeof(block_q5_1) == 2 * sizeof(ggml_fp16_t) + sizeof(uint32_t) + QK5_1 / 2, "wrong q5_1 block size/padding");

#define QK8_0 32
#define QR8_0 1
typedef struct {
    half    d;              // delta
    int8_t  qs[QK8_0];      // quants
} block_q8_0;
static_assert(sizeof(block_q8_0) == sizeof(ggml_fp16_t) + QK8_0, "wrong q8_0 block size/padding");

#define WARP_SIZE 32

#define CUDA_ADD_BLOCK_SIZE 256
#define CUDA_MUL_BLOCK_SIZE 256
#define CUDA_SILU_BLOCK_SIZE 256
#define CUDA_DEQUANTIZE_BLOCK_SIZE 256

// dmmv = dequantize_mul_mat_vec
#ifndef GGML_CUDA_DMMV_X
#define GGML_CUDA_DMMV_X 32
#endif
#ifndef GGML_CUDA_DMMV_Y
#define GGML_CUDA_DMMV_Y 1
#endif

static __global__ void add_f32(const float * x, const float * y, float * dst, const int k) {
    const int i = blockDim.x*blockIdx.x + threadIdx.x;

    if (i >= k) {
        return;
    }
    dst[i] = x[i] + y[i];
}

static __global__ void mul_f32(const float * x, const float * y, float * dst, const int kx, const int ky) {
    const int i = blockDim.x*blockIdx.x + threadIdx.x;

    if (i >= kx) {
        return;
    }
    dst[i] = x[i] * y[i%ky];
}

static __global__ void silu_f32(const float * x, float * dst, const int k) {
    const int i = blockDim.x*blockIdx.x + threadIdx.x;

    if (i >= k) {
        return;
    }
    dst[i] = x[i] / (1.0f + expf(-x[i]));
}

static __global__ void rms_norm_f32(const float * x, float * dst, const int ncols) {
    const int row = blockIdx.x*blockDim.y + threadIdx.y;
    const int tid = threadIdx.x;

    const float eps = 1e-6;

    float tmp = 0.0f; // partial sum for thread in warp

    for (int i = 0; i < ncols; i += WARP_SIZE) {
        const int col = i + tid;
        const float xi = x[row*ncols + col];
        tmp += xi * xi;
    }

    // sum up partial sums
    __syncthreads();
#pragma unroll
    for (int mask = 16; mask > 0; mask >>= 1) {
        tmp += __shfl_xor_sync(0xffffffff, tmp, mask, 32);
    }

    const float mean = tmp / ncols;
    const float scale = 1.0f / sqrtf(mean + eps);

    for (int i = 0; i < ncols; i += WARP_SIZE) {
        const int col = i + tid;
        dst[row*ncols + col] = scale * x[row*ncols + col];
    }
}

static __device__ void dequantize_q4_0(const void * vx, const int ib, const int iqs, float & v0, float & v1){
    const block_q4_0 * x = (const block_q4_0 *) vx;

    const float d = x[ib].d;

    const uint8_t vui = x[ib].qs[iqs];

    const int8_t vi0 = vui & 0xF;
    const int8_t vi1 = vui >> 4;

    v0 = (vi0 - 8)*d;
    v1 = (vi1 - 8)*d;
}

static __device__ void dequantize_q4_1(const void * vx, const int ib, const int iqs, float & v0, float & v1){
    const block_q4_1 * x = (const block_q4_1 *) vx;

    const float d = x[ib].d;
    const float m = x[ib].m;

    const uint8_t vui = x[ib].qs[iqs];

    const int8_t vi0 = vui & 0xF;
    const int8_t vi1 = vui >> 4;

    v0 = vi0*d + m;
    v1 = vi1*d + m;
}

static __device__ void dequantize_q5_0(const void * vx, const int ib, const int iqs, float & v0, float & v1){
    const block_q5_0 * x = (const block_q5_0 *) vx;

    const float d = x[ib].d;

    uint32_t qh;
    memcpy(&qh, x[ib].qh, sizeof(qh));

    const uint8_t xh_0 = ((qh >> (iqs +  0)) << 4) & 0x10;
    const uint8_t xh_1 = ((qh >> (iqs + 12))     ) & 0x10;

    const int32_t x0 = ((x[ib].qs[iqs] & 0xf) | xh_0) - 16;
    const int32_t x1 = ((x[ib].qs[iqs] >>  4) | xh_1) - 16;

    v0 = x0*d;
    v1 = x1*d;
}

static __device__ void dequantize_q5_1(const void * vx, const int ib, const int iqs, float & v0, float & v1){
    const block_q5_1 * x = (const block_q5_1 *) vx;

    const float d = x[ib].d;
    const float m = x[ib].m;

    uint32_t qh;
    memcpy(&qh, x[ib].qh, sizeof(qh));

    const uint8_t xh_0 = ((qh >> (iqs +  0)) << 4) & 0x10;
    const uint8_t xh_1 = ((qh >> (iqs + 12))     ) & 0x10;

    const int32_t x0 = ((x[ib].qs[iqs] & 0xf) | xh_0);
    const int32_t x1 = ((x[ib].qs[iqs] >>  4) | xh_1);

    v0 = x0*d + m;
    v1 = x1*d + m;
}

static __device__ void dequantize_q8_0(const void * vx, const int ib, const int iqs, float & v0, float & v1){
    const block_q8_0 * x = (const block_q8_0 *) vx;

    const float d = x[ib].d;

    const int8_t vi0 = x[ib].qs[iqs + 0];
    const int8_t vi1 = x[ib].qs[iqs + 1];

    v0 = vi0*d;
    v1 = vi1*d;
}

static __device__ void convert_f16(const void * vx, const int ib, const int iqs, float & v0, float & v1){
    const half * x = (const half *) vx;

    v0 = __half2float(x[ib + iqs + 0]);
    v1 = __half2float(x[ib + iqs + 1]);
}

template <int qk, int qr, dequantize_kernel_t dequantize_kernel>
static __global__ void dequantize_block(const void * vx, float * y, const int k) {
    const int i = blockDim.x*blockIdx.x + 2*threadIdx.x;

    if (i >= k) {
        return;
    }

    const int ib = i/qk; // block index
    const int iqs = (i%qk)/qr; // quant index
    const int iybs = i - i%qk; // y block start index
    const int y_offset = qr == 1 ? 1 : qk/2;

    // dequantize
    float & v0 = y[iybs + iqs + 0];
    float & v1 = y[iybs + iqs + y_offset];
    dequantize_kernel(vx, ib, iqs, v0, v1);
}

template <int qk, int qr, dequantize_kernel_t dequantize_kernel>
static __global__ void dequantize_mul_mat_vec(const void * vx, const float * y, float * dst, const int ncols) {
    // qk = quantized weights per x block
    // qr = number of quantized weights per data value in x block
    const int row = blockIdx.x*blockDim.y + threadIdx.y;
    const int tid = threadIdx.x;

    const int iter_stride = 2*GGML_CUDA_DMMV_X;
    const int vals_per_iter = iter_stride / WARP_SIZE; // num quantized vals per thread and i iter
    const int y_offset = qr == 1 ? 1 : qk/2;

    float tmp = 0.0f; // partial sum for thread in warp

    for (int i = 0; i < ncols; i += iter_stride) {
        const int col = i + vals_per_iter*tid;
        const int ib = (row*ncols + col)/qk; // x block index
        const int iqs = (col%qk)/qr; // x quant index
        const int iybs = col - col%qk; // y block start index

// processing >2 values per i iter is faster for fast GPUs
#pragma unroll
        for (int j = 0; j < vals_per_iter; j += 2) {
            // process 2 vals per j iter

            // dequantize
            float v0, v1;
            dequantize_kernel(vx, ib, iqs + j/qr, v0, v1);
            // for qr = 2 the iqs needs to increase by 1 per j iter because 2 weights per data val

            // matrix multiplication
            tmp += v0 * y[iybs + iqs + j/qr + 0];
            tmp += v1 * y[iybs + iqs + j/qr + y_offset];
            // for qr = 2 the y index needs to increase by 1 per j iter because of y_offset = qk/2
        }
    }

    // sum up partial sums and write back result
    __syncthreads();
#pragma unroll
    for (int mask = 16; mask > 0; mask >>= 1) {
        tmp += __shfl_xor_sync(0xffffffff, tmp, mask, 32);
    }

    if (tid == 0) {
        dst[row] = tmp;
    }
}

static void add_f32_cuda(const float * x, const float * y, float * dst, const int k, hipStream_t stream) {
    const int num_blocks = (k + CUDA_ADD_BLOCK_SIZE - 1) / CUDA_ADD_BLOCK_SIZE;
    add_f32<<<num_blocks, CUDA_ADD_BLOCK_SIZE, 0, stream>>>(x, y, dst, k);
}

static void mul_f32_cuda(const float * x, const float * y, float * dst, const int kx, const int ky, hipStream_t stream) {
    const int num_blocks = (kx + CUDA_MUL_BLOCK_SIZE - 1) / CUDA_MUL_BLOCK_SIZE;
    mul_f32<<<num_blocks, CUDA_MUL_BLOCK_SIZE, 0, stream>>>(x, y, dst, kx, ky);
}

static void silu_f32_cuda(const float * x, float * dst, const int k, hipStream_t stream) {
    const int num_blocks = (k + CUDA_SILU_BLOCK_SIZE - 1) / CUDA_SILU_BLOCK_SIZE;
    silu_f32<<<num_blocks, CUDA_SILU_BLOCK_SIZE, 0, stream>>>(x, dst, k);
}

static void rms_norm_f32_cuda(const float * x, float * dst, const int ncols, const int nrows, hipStream_t stream) {
    GGML_ASSERT(ncols % WARP_SIZE == 0);
    const dim3 block_dims(WARP_SIZE, 1, 1);
    rms_norm_f32<<<nrows, block_dims, 0, stream>>>(x, dst, ncols);
}

static void dequantize_row_q4_0_cuda(const void * vx, float * y, const int k, hipStream_t stream) {
    const int num_blocks = (k + CUDA_DEQUANTIZE_BLOCK_SIZE - 1) / CUDA_DEQUANTIZE_BLOCK_SIZE;
    dequantize_block<QK4_0, QR4_0, dequantize_q4_0><<<num_blocks, CUDA_DEQUANTIZE_BLOCK_SIZE, 0, stream>>>(vx, y, k);
}

static void dequantize_row_q4_1_cuda(const void * vx, float * y, const int k, hipStream_t stream) {
    const int num_blocks = (k + CUDA_DEQUANTIZE_BLOCK_SIZE - 1) / CUDA_DEQUANTIZE_BLOCK_SIZE;
    dequantize_block<QK4_1, QR4_1, dequantize_q4_1><<<num_blocks, CUDA_DEQUANTIZE_BLOCK_SIZE, 0, stream>>>(vx, y, k);
}

static void dequantize_row_q5_0_cuda(const void * vx, float * y, const int k, hipStream_t stream) {
    const int num_blocks = (k + CUDA_DEQUANTIZE_BLOCK_SIZE - 1) / CUDA_DEQUANTIZE_BLOCK_SIZE;
    dequantize_block<QK5_0, QR5_0, dequantize_q5_0><<<num_blocks, CUDA_DEQUANTIZE_BLOCK_SIZE, 0, stream>>>(vx, y, k);
}

static void dequantize_row_q5_1_cuda(const void * vx, float * y, const int k, hipStream_t stream) {
    const int num_blocks = (k + CUDA_DEQUANTIZE_BLOCK_SIZE - 1) / CUDA_DEQUANTIZE_BLOCK_SIZE;
    dequantize_block<QK5_1, QR5_1, dequantize_q5_1><<<num_blocks, CUDA_DEQUANTIZE_BLOCK_SIZE, 0, stream>>>(vx, y, k);
}

static void dequantize_row_q8_0_cuda(const void * vx, float * y, const int k, hipStream_t stream) {
    const int num_blocks = (k + CUDA_DEQUANTIZE_BLOCK_SIZE - 1) / CUDA_DEQUANTIZE_BLOCK_SIZE;
    dequantize_block<QK8_0, QR8_0, dequantize_q8_0><<<num_blocks, CUDA_DEQUANTIZE_BLOCK_SIZE, 0, stream>>>(vx, y, k);
}

static void dequantize_mul_mat_vec_q4_0_cuda(const void * vx, const float * y, float * dst, const int ncols, const int nrows, hipStream_t stream) {
    GGML_ASSERT(ncols % GGML_CUDA_DMMV_X == 0);
    GGML_ASSERT(nrows % GGML_CUDA_DMMV_Y == 0);
    const dim3 block_dims(WARP_SIZE, GGML_CUDA_DMMV_Y, 1);
    dequantize_mul_mat_vec<QK4_0, QR4_0, dequantize_q4_0>
        <<<nrows/GGML_CUDA_DMMV_Y, block_dims, 0, stream>>>(vx, y, dst, ncols);
}

static void dequantize_mul_mat_vec_q4_1_cuda(const void * vx, const float * y, float * dst, const int ncols, const int nrows, hipStream_t stream) {
    GGML_ASSERT(ncols % GGML_CUDA_DMMV_X == 0);
    GGML_ASSERT(nrows % GGML_CUDA_DMMV_Y == 0);
    const dim3 block_dims(WARP_SIZE, GGML_CUDA_DMMV_Y, 1);
    dequantize_mul_mat_vec<QK4_1, QR4_1, dequantize_q4_1>
        <<<nrows/GGML_CUDA_DMMV_Y, block_dims, 0, stream>>>(vx, y, dst, ncols);
}

static void dequantize_mul_mat_vec_q5_0_cuda(const void * vx, const float * y, float * dst, const int ncols, const int nrows, hipStream_t stream) {
    GGML_ASSERT(ncols % GGML_CUDA_DMMV_X == 0);
    GGML_ASSERT(nrows % GGML_CUDA_DMMV_Y == 0);
    const dim3 block_dims(WARP_SIZE, GGML_CUDA_DMMV_Y, 1);
    dequantize_mul_mat_vec<QK5_0, QR5_0, dequantize_q5_0>
        <<<nrows/GGML_CUDA_DMMV_Y, block_dims, 0, stream>>>(vx, y, dst, ncols);
}

static void dequantize_mul_mat_vec_q5_1_cuda(const void * vx, const float * y, float * dst, const int ncols, const int nrows, hipStream_t stream) {
    GGML_ASSERT(ncols % GGML_CUDA_DMMV_X == 0);
    GGML_ASSERT(nrows % GGML_CUDA_DMMV_Y == 0);
    const dim3 block_dims(WARP_SIZE, GGML_CUDA_DMMV_Y, 1);
    dequantize_mul_mat_vec<QK5_1, QR5_1, dequantize_q5_1>
        <<<nrows/GGML_CUDA_DMMV_Y, block_dims, 0, stream>>>(vx, y, dst, ncols);
}

static void dequantize_mul_mat_vec_q8_0_cuda(const void * vx, const float * y, float * dst, const int ncols, const int nrows, hipStream_t stream) {
    GGML_ASSERT(ncols % GGML_CUDA_DMMV_X == 0);
    GGML_ASSERT(nrows % GGML_CUDA_DMMV_Y == 0);
    const dim3 block_dims(WARP_SIZE, GGML_CUDA_DMMV_Y, 1);
    dequantize_mul_mat_vec<QK8_0, QR8_0, dequantize_q8_0>
        <<<nrows/GGML_CUDA_DMMV_Y, block_dims, 0, stream>>>(vx, y, dst, ncols);
}

static void convert_fp16_to_fp32_cuda(const void * vx, float * y, const int k, hipStream_t stream) {
    const int num_blocks = (k + CUDA_DEQUANTIZE_BLOCK_SIZE - 1) / CUDA_DEQUANTIZE_BLOCK_SIZE;
    dequantize_block<1, 1, convert_f16><<<num_blocks, CUDA_DEQUANTIZE_BLOCK_SIZE, 0, stream>>>(vx, y, k);
}

static void convert_mul_mat_vec_f16_cuda(const void * vx, const float * y, float * dst, const int ncols, const int nrows, hipStream_t stream) {
    GGML_ASSERT(ncols % GGML_CUDA_DMMV_X == 0);
    GGML_ASSERT(nrows % GGML_CUDA_DMMV_Y == 0);
    const dim3 block_dims(WARP_SIZE, GGML_CUDA_DMMV_Y, 1);
    dequantize_mul_mat_vec<1, 1, convert_f16>
        <<<nrows/GGML_CUDA_DMMV_Y, block_dims, 0, stream>>>(vx, y, dst, ncols);
}

static to_fp32_cuda_t ggml_get_to_fp32_cuda(ggml_type type) {
    switch (type) {
        case GGML_TYPE_Q4_0:
            return dequantize_row_q4_0_cuda;
        case GGML_TYPE_Q4_1:
            return dequantize_row_q4_1_cuda;
        case GGML_TYPE_Q5_0:
            return dequantize_row_q5_0_cuda;
        case GGML_TYPE_Q5_1:
            return dequantize_row_q5_1_cuda;
        case GGML_TYPE_Q8_0:
            return dequantize_row_q8_0_cuda;
        case GGML_TYPE_F16:
            return convert_fp16_to_fp32_cuda;
        default:
            return nullptr;
    }
}

// buffer pool for cuda
#define MAX_CUDA_BUFFERS 256

struct scoped_spin_lock {
    std::atomic_flag& lock;
    scoped_spin_lock(std::atomic_flag& lock) : lock(lock) {
        while (lock.test_and_set(std::memory_order_acquire)) {
            ; // spin
        }
    }
    ~scoped_spin_lock() {
        lock.clear(std::memory_order_release);
    }
    scoped_spin_lock(const scoped_spin_lock&) = delete;
    scoped_spin_lock& operator=(const scoped_spin_lock&) = delete;
};

struct cuda_buffer {
    void * ptr = nullptr;
    size_t size = 0;
};

static cuda_buffer g_cuda_buffer_pool[GGML_CUDA_MAX_DEVICES][MAX_CUDA_BUFFERS];
static std::atomic_flag g_cuda_pool_lock = ATOMIC_FLAG_INIT;

static void * ggml_cuda_pool_malloc(size_t size, size_t * actual_size) {
    scoped_spin_lock lock(g_cuda_pool_lock);
    int id;
    CUDA_CHECK(hipGetDevice(&id));

    for (int i = 0; i < MAX_CUDA_BUFFERS; ++i) {
        cuda_buffer& b = g_cuda_buffer_pool[id][i];
        if (b.size >= size && b.ptr != nullptr) {
            void * ptr = b.ptr;
            *actual_size = b.size;
            b.ptr = nullptr;
            b.size = 0;
            return ptr;
        }
    }
    void * ptr;
    CUDA_CHECK(hipMalloc((void **) &ptr, size));
    *actual_size = size;
    return ptr;
}

static void ggml_cuda_pool_free(void * ptr, size_t size) {
    scoped_spin_lock lock(g_cuda_pool_lock);
    int id;
    CUDA_CHECK(hipGetDevice(&id));

    for (int i = 0; i < MAX_CUDA_BUFFERS; ++i) {
        cuda_buffer& b = g_cuda_buffer_pool[id][i];
        if (b.ptr == nullptr) {
            b.ptr = ptr;
            b.size = size;
            return;
        }
    }
    fprintf(stderr, "WARNING: cuda buffer pool full, increase MAX_CUDA_BUFFERS\n");
    CUDA_CHECK(hipFree(ptr));
}

#define GGML_CUDA_MAX_SCRATCH_BUFFERS 16
#define GGML_CUDA_SCRATCH_SIZE 536870912 // 512 MB
//#define GGML_CUDA_SCRATCH_SIZE 1073741824 // 1 GB
//#define GGML_CUDA_SCRATCH_SIZE 4294967296 // 4 GB
static void * g_scratch_buffers[GGML_CUDA_MAX_DEVICES][GGML_CUDA_MAX_SCRATCH_BUFFERS] = {nullptr};
static int g_scratch_index = 0;
static size_t g_scratch_offset = 0;

#define GGML_CUDA_MAX_STREAMS 8 // Set this to 1 for reproducible matrix multiplication.
#define GGML_CUDA_MAX_EVENTS 64

static int g_device_count = -1;
static int g_main_device = 0;
static float g_tensor_split[GGML_CUDA_MAX_DEVICES] = {0};

static hipblasHandle_t g_cublas_handles[GGML_CUDA_MAX_DEVICES] = {nullptr};

static hipStream_t g_cudaStreams_main[GGML_CUDA_MAX_DEVICES][GGML_CUDA_MAX_STREAMS] = { nullptr };

static hipStream_t g_cudaStreams_memcpy_src1[GGML_CUDA_MAX_DEVICES][GGML_CUDA_MAX_STREAMS] = { nullptr };
static hipEvent_t g_cudaEvents_memcpy_src1[GGML_CUDA_MAX_DEVICES][GGML_CUDA_MAX_EVENTS] = { nullptr };

void ggml_init_cublas() {
    static bool initialized = false;

    if (!initialized) {
        CUDA_CHECK(hipGetDeviceCount(&g_device_count));
        GGML_ASSERT(g_device_count <= GGML_CUDA_MAX_DEVICES);
        int64_t total_vram = 0;
        fprintf(stderr, "%s: found %d CUDA devices:\n", __func__, g_device_count);
        for (int i = 0; i < g_device_count; ++i) {
            hipDeviceProp_t prop;
            CUDA_CHECK(hipGetDeviceProperties(&prop, i));
            fprintf(stderr, "  %d. %s\n", i+1, prop.name);
            g_tensor_split[i] = total_vram;
            total_vram += prop.totalGlobalMem;
        }
        for (int i = 0; i < g_device_count; ++i) {
            g_tensor_split[i] /= total_vram;
        }

        for (int id = 0; id < g_device_count; ++id) {
            CUDA_CHECK(hipSetDevice(id));

            // create streams
            for (int i = 0; i < GGML_CUDA_MAX_STREAMS; ++i) {
                CUDA_CHECK(hipStreamCreateWithFlags(&g_cudaStreams_main[id][i], hipStreamNonBlocking));
                CUDA_CHECK(hipStreamCreateWithFlags(&g_cudaStreams_memcpy_src1[id][i], hipStreamNonBlocking));
            }
            // create events
            for (int i = 0; i < GGML_CUDA_MAX_EVENTS; ++i) {
                CUDA_CHECK(hipEventCreateWithFlags(&g_cudaEvents_memcpy_src1[id][i], hipEventDisableTiming));
            }

            // create cublas handle
            CUBLAS_CHECK(hipblasCreate(&g_cublas_handles[id]));
            CUBLAS_CHECK(hipblasSetMathMode(g_cublas_handles[id], HIPBLAS_TF32_TENSOR_OP_MATH));
        }

        // configure logging to stdout
        // CUBLAS_CHECK(cublasLoggerConfigure(1, 1, 0, nullptr));

        initialized = true;
    }
}

void ggml_cuda_set_tensor_split(float * tensor_split) {
    bool all_zero = true;
    for (int i = 0; i < g_device_count; ++i) {
        if (tensor_split[i] != 0.0f) {
            all_zero = false;
            break;
        }
    }
    if (all_zero) {
        return;
    }
    float split_sum = 0.0f;
    for (int i = 0; i < g_device_count; ++i) {
        g_tensor_split[i] = split_sum;
        split_sum += tensor_split[i];
    }
    for (int i = 0; i < g_device_count; ++i) {
        g_tensor_split[i] /= split_sum;
    }
}

void * ggml_cuda_host_malloc(size_t size) {
    if (getenv("GGML_CUDA_NO_PINNED") != nullptr) {
        return nullptr;
    }

    void * ptr = nullptr;
    hipError_t err = hipHostMalloc((void **) &ptr, size);
    if (err != hipSuccess) {
        fprintf(stderr, "WARNING: failed to allocate %.2f MB of pinned memory: %s\n",
            size/1024.0/1024.0, hipGetErrorString(err));
        return nullptr;
    }

    return ptr;
}

void ggml_cuda_host_free(void * ptr) {
    CUDA_CHECK(hipHostFree(ptr));
}

static hipError_t ggml_cuda_h2d_tensor_2d(
    void * dst, const struct ggml_tensor * src, int64_t i3, int64_t i2, int64_t i1_low, int64_t i1_high, hipStream_t stream) {

    char * dst_char = (char *) dst;
    const int64_t ne0 = src->ne[0];
    const int64_t nb0 = src->nb[0];
    const int64_t nb1 = src->nb[1];
    const int64_t nb2 = src->nb[2];
    const int64_t nb3 = src->nb[3];
    const enum ggml_type type = src->type;
    const int64_t ts = ggml_type_size(type);
    const int64_t bs = ggml_blck_size(type);
    int64_t i1_diff = i1_high - i1_low;

    const void * x = (const void *) ((const char *) src->data + i1_low*nb1 + i2*nb2 + i3*nb3);
    if (nb0 == ts && nb1 == ts*ne0/bs) {
        return hipMemcpyAsync(dst_char, x, i1_diff*nb1, hipMemcpyHostToDevice, stream);
    } else if (nb0 == ts) {
        return hipMemcpy2DAsync(dst_char, ts*ne0/bs, x, nb1, ts*ne0/bs, i1_diff, hipMemcpyHostToDevice, stream);
    } else {
        for (int64_t i1 = 0; i1 < i1_diff; i1++) {
            const void * rx = (const void *) ((const char *) x + i1*nb1);
            void * rd = (void *) (dst_char + i1*ts*ne0/bs);
            // pretend the row is a matrix with cols=1
            hipError_t r = hipMemcpy2DAsync(rd, ts/bs, rx, nb0, ts/bs, ne0, hipMemcpyHostToDevice, stream);
            if (r != hipSuccess) return r;
        }
        return hipSuccess;
    }
}

inline void ggml_cuda_op_add(
    const ggml_tensor * src0, const ggml_tensor * src1, ggml_tensor * dst, char * src0_ddq_i,
    float * src0_ddf_i, float * src1_ddf_i, float * dst_ddf_i, int64_t i0_low, int64_t i0_high, int i1,
    hipStream_t & cudaStream_main){

    GGML_ASSERT(src0_ddf_i != nullptr);
    GGML_ASSERT(src1_ddf_i != nullptr);
    GGML_ASSERT(dst_ddf_i != nullptr);

    const int64_t ne0 = src0->ne[0];
    const int64_t i0_diff = i0_high - i0_low;

    // compute
    add_f32_cuda(src0_ddf_i, src1_ddf_i, dst_ddf_i, ne0*i0_diff, cudaStream_main);
    CUDA_CHECK(hipGetLastError());

    (void) src1;
    (void) dst;
    (void) src0_ddq_i;
    (void) i1;
}

inline void ggml_cuda_op_mul(
    const ggml_tensor * src0, const ggml_tensor * src1, ggml_tensor * dst, char * src0_ddq_i,
    float * src0_ddf_i, float * src1_ddf_i, float * dst_ddf_i, int64_t i0_low, int64_t i0_high, int i1,
    hipStream_t & cudaStream_main){

    GGML_ASSERT(src0_ddf_i != nullptr);
    GGML_ASSERT(src1_ddf_i != nullptr);
    GGML_ASSERT(dst_ddf_i != nullptr);

    const int64_t ne00 = src0->ne[0];

    const int64_t ne10 = src1->ne[0];
    const int64_t ne11 = src1->ne[1];

    for (int64_t i01 = i0_low; i01 < i0_high; i01++) {
        const int64_t i11 = i1*ne11 + i01%ne11; // broadcast src1 across src0

        float * src0_ddf_i01 = src0_ddf_i + i01*ne00;
        float * src1_ddf_i01 = src1_ddf_i + i11*ne10;
        float * dst_ddf_i01 = dst_ddf_i + i01*ne00;

        // compute
        mul_f32_cuda(src0_ddf_i01, src1_ddf_i01, dst_ddf_i01, ne00, ne10, cudaStream_main);
        CUDA_CHECK(hipGetLastError());
    }

    (void) dst;
    (void) src0_ddq_i;
}

inline void ggml_cuda_op_silu(
    const ggml_tensor * src0, const ggml_tensor * src1, ggml_tensor * dst, char * src0_ddq_i,
    float * src0_ddf_i, float * src1_ddf_i, float * dst_ddf_i, int64_t i0_low, int64_t i0_high, int i1,
    hipStream_t & cudaStream_main){

    GGML_ASSERT(src0_ddf_i != nullptr);
    GGML_ASSERT(dst_ddf_i != nullptr);

    const int64_t ne00 = src0->ne[0];
    const int64_t i0_diff = i0_high - i0_low;

    // compute
    silu_f32_cuda(src0_ddf_i, dst_ddf_i, ne00*i0_diff, cudaStream_main);
    CUDA_CHECK(hipGetLastError());

    (void) src1;
    (void) dst;
    (void) src0_ddq_i;
    (void) src1_ddf_i;
    (void) i1;
}

inline void ggml_cuda_op_rms_norm(
    const ggml_tensor * src0, const ggml_tensor * src1, ggml_tensor * dst, char * src0_ddq_i,
    float * src0_ddf_i, float * src1_ddf_i, float * dst_ddf_i, int64_t i0_low, int64_t i0_high, int i1,
    hipStream_t & cudaStream_main){

    GGML_ASSERT(src0_ddf_i != nullptr);
    GGML_ASSERT(dst_ddf_i != nullptr);

    const int64_t ne00 = src0->ne[0];
    const int64_t i0_diff = i0_high - i0_low;

    // compute
    rms_norm_f32_cuda(src0_ddf_i, dst_ddf_i, ne00, i0_diff, cudaStream_main);
    CUDA_CHECK(hipGetLastError());

    (void) src1;
    (void) dst;
    (void) src0_ddq_i;
    (void) src1_ddf_i;
    (void) i1;
}

inline void ggml_cuda_op_dequantize_mul_mat_vec(
    const ggml_tensor * src0, const ggml_tensor * src1, ggml_tensor * dst, char * src0_ddq_i,
    float * src0_ddf_i, float * src1_ddf_i, float * dst_ddf_i, int64_t i0_low, int64_t i0_high, int i1,
    hipStream_t & cudaStream_main){

    GGML_ASSERT(src0_ddq_i != nullptr);
    GGML_ASSERT(src1_ddf_i != nullptr);
    GGML_ASSERT(dst_ddf_i != nullptr);

    const int64_t ne00 = src0->ne[0];
    const int64_t nrows = i0_high - i0_low;

    switch (src0->type) {
        case GGML_TYPE_Q4_0:
            dequantize_mul_mat_vec_q4_0_cuda(src0_ddq_i, src1_ddf_i, dst_ddf_i, ne00, nrows, cudaStream_main);
            break;
        case GGML_TYPE_Q4_1:
            dequantize_mul_mat_vec_q4_1_cuda(src0_ddq_i, src1_ddf_i, dst_ddf_i, ne00, nrows, cudaStream_main);
            break;
        case GGML_TYPE_Q5_0:
            dequantize_mul_mat_vec_q5_0_cuda(src0_ddq_i, src1_ddf_i, dst_ddf_i, ne00, nrows, cudaStream_main);
            break;
        case GGML_TYPE_Q5_1:
            dequantize_mul_mat_vec_q5_1_cuda(src0_ddq_i, src1_ddf_i, dst_ddf_i, ne00, nrows, cudaStream_main);
            break;
        case GGML_TYPE_Q8_0:
            dequantize_mul_mat_vec_q8_0_cuda(src0_ddq_i, src1_ddf_i, dst_ddf_i, ne00, nrows, cudaStream_main);
            break;
        case GGML_TYPE_F16:
            convert_mul_mat_vec_f16_cuda(src0_ddq_i, src1_ddf_i, dst_ddf_i, ne00, nrows, cudaStream_main);
            break;
        default:
            GGML_ASSERT(false);
            break;
    }
    CUDA_CHECK(hipGetLastError());

    (void) src1;
    (void) dst;
    (void) src0_ddf_i;
    (void) i1;
}

inline void ggml_cuda_op_mul_mat_cublas(
    const ggml_tensor * src0, const ggml_tensor * src1, ggml_tensor * dst, char * src0_ddq_i,
    float * src0_ddf_i, float * src1_ddf_i, float * dst_ddf_i, int64_t i0_low, int64_t i0_high, int i1,
    hipStream_t & cudaStream_main){

    GGML_ASSERT(src0_ddf_i != nullptr);
    GGML_ASSERT(src1_ddf_i != nullptr);
    GGML_ASSERT(dst_ddf_i != nullptr);

    const float alpha = 1.0f;
    const float beta = 0.0f;

    const int64_t ne00 = src0->ne[0];

    const int64_t ne10 = src1->ne[0];
    const int64_t ne11 = src1->ne[1];

    const int64_t i0_diff = i0_high - i0_low;

    int id;
    CUDA_CHECK(hipGetDevice(&id));

    CUBLAS_CHECK(hipblasSetStream(g_cublas_handles[id], cudaStream_main));
    CUBLAS_CHECK(
        hipblasSgemm(g_cublas_handles[id], HIPBLAS_OP_T, HIPBLAS_OP_N,
                i0_diff, ne11, ne10,
                &alpha, src0_ddf_i, ne00,
                        src1_ddf_i, ne10,
                &beta,  dst_ddf_i,  i0_diff));

    (void) dst;
    (void) src0_ddq_i;
    (void) i1;
}

static void ggml_cuda_op(const ggml_tensor * src0, const ggml_tensor * src1, ggml_tensor * dst,
                         ggml_cuda_op_t op, bool src0_needs_f32) {
    const int64_t ne00 = src0->ne[0];
    const int64_t ne01 = src0->ne[1];
    const int64_t ne02 = src0->ne[2];
    const int64_t ne03 = src0->ne[3];
    const int64_t nrows0 = ggml_nrows(src0);

    const bool use_src1 = src1 != nullptr;
    const int64_t ne10 = use_src1 ? src1->ne[0] : 1;
    const int64_t ne11 = use_src1 ? src1->ne[1] : 1;
    const int64_t ne12 = use_src1 ? src1->ne[2] : 1;
    const int64_t ne13 = use_src1 ? src1->ne[3] : 1;

    const int64_t ne0 = dst->ne[0];
    const int64_t ne1 = dst->ne[1];

    const int nb2  = dst->nb[2];
    const int nb3  = dst->nb[3];

    GGML_ASSERT(dst->backend != GGML_BACKEND_GPU_SPLIT);
    GGML_ASSERT(!use_src1 || src1->backend != GGML_BACKEND_GPU_SPLIT);

    // strides for iteration over dims 3 and 2
    const int64_t src0_stride = ne00 * ne01;
    const int64_t src1_stride = ne10 * ne11;
    const int64_t dst_stride = ne0 * ne1;
    const int64_t num_iters = ne02 * ne03;

    const size_t src0_ts = ggml_type_size(src0->type);
    const size_t src0_bs = ggml_blck_size(src0->type);

    struct ggml_tensor_extra_gpu * src0_extra = (ggml_tensor_extra_gpu *) src0->extra;
    struct ggml_tensor_extra_gpu * src1_extra = use_src1 ? (ggml_tensor_extra_gpu *) src1->extra : nullptr;
    struct ggml_tensor_extra_gpu * dst_extra = (ggml_tensor_extra_gpu *) dst->extra;

    const bool src0_on_device = src0->backend == GGML_BACKEND_GPU || src0->backend == GGML_BACKEND_GPU_SPLIT;
    const bool src0_is_f32 = src0->type == GGML_TYPE_F32;

    const bool src1_on_device = use_src1 && src1->backend == GGML_BACKEND_GPU;

    const bool dst_on_device = dst->backend == GGML_BACKEND_GPU;

    const bool split = src0->backend == GGML_BACKEND_GPU_SPLIT;

    const to_fp32_cuda_t to_fp32_cuda = ggml_get_to_fp32_cuda(src0->type);

    // dd = data device
    char  * src0_ddq[GGML_CUDA_MAX_DEVICES] = {nullptr}; // quantized
    float * src0_ddf[GGML_CUDA_MAX_DEVICES] = {nullptr}; // float
    float * src1_ddf[GGML_CUDA_MAX_DEVICES] = {nullptr};
    float * dst_ddf[GGML_CUDA_MAX_DEVICES] = {nullptr};

    // asq = actual size quantized, asf = actual size float
    size_t src0_asq[GGML_CUDA_MAX_DEVICES] = {0};
    size_t src0_asf[GGML_CUDA_MAX_DEVICES] = {0};
    size_t src1_asf[GGML_CUDA_MAX_DEVICES] = {0};
    size_t dst_asf[GGML_CUDA_MAX_DEVICES] = {0};

    for (int id = 0; id < g_device_count; ++id) {
        if (!split && id != g_main_device) {
            continue;
        }

        int64_t row_low, row_high;
        if (split) {
            row_low = id == 0 ? 0 : nrows0*g_tensor_split[id];
            row_low -= row_low % GGML_CUDA_DMMV_Y;
            row_high = id == g_device_count - 1 ? nrows0 : nrows0*g_tensor_split[id + 1];
            row_high -= row_high % GGML_CUDA_DMMV_Y;
        } else {
            row_low = 0;
            row_high = ne01;
        }
        if (row_low == row_high) {
            continue;
        }

        int64_t row_diff = row_high - row_low;

        hipSetDevice(id);

        if (src0_on_device) {
            if (src0_is_f32) {
                src0_ddf[id] = (float *) src0_extra->data_device[id];
            } else {
                src0_ddq[id] = (char *) src0_extra->data_device[id];
            }
        } else {
            if (src0_is_f32) {
                src0_ddf[id] = (float *) ggml_cuda_pool_malloc(row_diff*ne00 * sizeof(float), &src0_asf[id]);
            } else {
                src0_ddq[id] = (char *) ggml_cuda_pool_malloc(row_diff*ne00 * src0_ts/src0_bs, &src0_asq[id]);
            }
        }

        if (src0_needs_f32 && !src0_is_f32) {
            src0_ddf[id] = (float *) ggml_cuda_pool_malloc(row_diff*ne00 * sizeof(float), &src0_asf[id]);
        }

        if (use_src1 && src1->backend == GGML_BACKEND_GPU) {
            src1_ddf[id] = (float *) src1_extra->data_device[id];
        } else {
            src1_ddf[id] = (float *) ggml_cuda_pool_malloc(num_iters*src1_stride * sizeof(float), &src1_asf[id]);
        }
        if (dst_on_device) {
            dst_ddf[id] = (float *) dst_extra->data_device[id];
        } else {
            size_t size_dst_ddf = split ? row_diff*ne1 * sizeof(float) : num_iters*dst_stride * sizeof(float);
            dst_ddf[id] = (float *) ggml_cuda_pool_malloc(size_dst_ddf, &dst_asf[id]);
        }

        for (int64_t i03 = 0; i03 < ne03; i03++) {
            const int64_t i13 = i03 % ne13;
            for (int64_t i02 = 0; i02 < ne02; i02++) {
                const int64_t i12 = i02 % ne12;

                const int64_t i0 = i03*ne02 + i02;
                const int64_t i0_offset_low = row_low/ne01;
                const int64_t i0_offset_high = row_high/ne01;

                int64_t i01_low = 0;
                int64_t i01_high = ne01;
                if (split) {
                    if (i0 < i0_offset_low || i0 > i0_offset_high) {
                        continue;
                    }
                    if (i0 == i0_offset_low) {
                        i01_low = row_low % ne01;
                    }
                    if (i0 == i0_offset_high) {
                        i01_high = row_high % ne01;
                    }
                }
                const int64_t i01_diff = i01_high - i01_low;
                if (i01_diff == 0) {
                    continue;
                }
                const int64_t i1 = i13*ne12 + i12;

                hipStream_t cudaStream_main = g_cudaStreams_main[id][i0 % GGML_CUDA_MAX_STREAMS];
                hipStream_t cudaStream_memcpy_src1 = g_cudaStreams_memcpy_src1[id][i0 % GGML_CUDA_MAX_STREAMS];
                hipEvent_t  cudaEvent_memcpy_src1 = g_cudaEvents_memcpy_src1[id][i0 % GGML_CUDA_MAX_EVENTS];

                // for split tensors the data begins at i0 == i0_offset_low
                char  * src0_ddq_i = src0_ddq[id] + (i0 - i0_offset_low)*src0_stride*src0_ts/src0_bs;
                float * src0_ddf_i = src0_ddf[id] + (i0 - i0_offset_low)*src0_stride;
                float * src1_ddf_i = src1_ddf[id] + i1*src1_stride;
                float * dst_ddf_i = dst_ddf[id] + (i0 - i0_offset_low)*dst_stride;

                // for split tensors the data pointer needs to be rounded down
                // to the bin edge for i03, i02 bins beyond the first
                if (i0 - i0_offset_low > 0) {
                    src0_ddq_i -= (row_low % ne01)*ne00 * src0_ts/src0_bs;
                    src0_ddf_i -= (row_low % ne01)*ne00;
                }
                if (i0 - i0_offset_low > 0) {
                    dst_ddf_i -= (row_low % ne0)*ne1;
                }

                // copy src0, src1 to device if necessary
                if (use_src1 && !src1_on_device) {
                    if (src1->backend == GGML_BACKEND_CPU) {
                        CUDA_CHECK(ggml_cuda_h2d_tensor_2d(src1_ddf_i, src1, i03, i02, 0, ne11, cudaStream_memcpy_src1));
                    } else if (src1->backend == GGML_BACKEND_GPU) {
                        float * src1_ddf_i_source = (float *) src1_extra->data_device[g_main_device];
                        src1_ddf_i_source += i1*src1_stride;
                        CUDA_CHECK(hipMemcpyAsync(src1_ddf_i, src1_ddf_i_source, src1_stride*sizeof(float),
                                                   hipMemcpyDeviceToDevice, cudaStream_memcpy_src1));
                    } else {
                        GGML_ASSERT(false);
                    }
                }
                CUDA_CHECK(hipEventRecord(cudaEvent_memcpy_src1, cudaStream_memcpy_src1));
                if (!src0_on_device) {
                    if (src0_is_f32) {
                        CUDA_CHECK(ggml_cuda_h2d_tensor_2d(src0_ddf_i, src0, i03, i02, i01_low, i01_high, cudaStream_main));
                    } else {
                        CUDA_CHECK(ggml_cuda_h2d_tensor_2d(src0_ddq_i, src0, i03, i02, i01_low, i01_high, cudaStream_main));
                    }
                }

                // convert src0 to f32 if it's necessary for the ggml_cuda_op
                if (src0_needs_f32 && !src0_is_f32) {
                    to_fp32_cuda(src0_ddq_i, src0_ddf_i, i01_diff*ne00, cudaStream_main);
                    CUDA_CHECK(hipGetLastError());
                }

                // wait with main stream until src1 memcpy is done
                CUDA_CHECK(hipStreamWaitEvent(cudaStream_main, cudaEvent_memcpy_src1, 0));

                // do the computation
                op(src0, src1, dst, src0_ddq_i, src0_ddf_i, src1_ddf_i, dst_ddf_i, i01_low, i01_high, i1, cudaStream_main);

                // copy dst to host or other device if necessary
                if (!dst_on_device) {
                    void * dst_off_device;
                    hipMemcpyKind kind;
                    if (dst->backend == GGML_BACKEND_CPU) {
                        dst_off_device = dst->data;
                        kind = hipMemcpyDeviceToHost;
                    } else if (dst->backend == GGML_BACKEND_GPU) {
                        dst_off_device = dst_extra->data_device[g_main_device];
                        kind = hipMemcpyDeviceToDevice;
                    } else {
                        GGML_ASSERT(false);
                    }
                    if (split) {
                        // src0 = weight matrix is saved as a transposed matrix for better memory layout.
                        // dst is NOT transposed.
                        // The outputs of cuBLAS matrix matrix multiplications can therefore NOT simply be concatenated for >1 GPU.
                        // Instead they need to be copied to the correct slice in ne0 = dst row index.
                        // If dst is a vector with ne0 == 1 then you don't have to do this but it still produces correct results.
                        for (int64_t j = 0; j < ne1; ++j) {
                            float * dhf_dst_i = (float *) ((char *) dst_off_device + (j*ne0 + i01_low)*sizeof(float) + i02*nb2 + i03*nb3);
                            CUDA_CHECK(hipMemcpyAsync(dhf_dst_i, dst_ddf_i + j*i01_diff, i01_diff*sizeof(float), kind, cudaStream_main));
                        }
                    } else {
                        float * dhf_dst_i = (float *) ((char *) dst_off_device + i02*nb2 + i03*nb3);
                        CUDA_CHECK(hipMemcpyAsync(dhf_dst_i, dst_ddf_i, dst_stride*sizeof(float), kind, cudaStream_main));
                    }
                }
            }
        }
    }

    // wait until each device is finished, then free their buffers
    for (int id = 0; id < g_device_count; ++id) {
        CUDA_CHECK(hipSetDevice(id));
        CUDA_CHECK(hipDeviceSynchronize());
        if (src0_asq[id] > 0) {
            ggml_cuda_pool_free(src0_ddq[id], src0_asq[id]);
        }
        if (src0_asf[id] > 0) {
            ggml_cuda_pool_free(src0_ddf[id], src0_asf[id]);
        }
        if (src1_asf[id] > 0) {
            ggml_cuda_pool_free(src1_ddf[id], src1_asf[id]);
        }
        if (dst_asf[id] > 0) {
            ggml_cuda_pool_free(dst_ddf[id], dst_asf[id]);
        }
    }
}

void ggml_cuda_add(const ggml_tensor * src0, const ggml_tensor * src1, ggml_tensor * dst) {
    GGML_ASSERT(src0->type == GGML_TYPE_F32 && src1->type == GGML_TYPE_F32 && dst->type == GGML_TYPE_F32);
    ggml_cuda_op(src0, src1, dst, ggml_cuda_op_add, true);
}

void ggml_cuda_mul(const ggml_tensor * src0, const ggml_tensor * src1, ggml_tensor * dst) {
    GGML_ASSERT(src0->type == GGML_TYPE_F32 && src1->type == GGML_TYPE_F32 && dst->type == GGML_TYPE_F32);
    ggml_cuda_op(src0, src1, dst, ggml_cuda_op_mul, true);
}

void ggml_cuda_silu(const ggml_tensor * src0, const ggml_tensor * src1, ggml_tensor * dst) {
    GGML_ASSERT(src0->type == GGML_TYPE_F32 && dst->type == GGML_TYPE_F32);
    ggml_cuda_op(src0, src1, dst, ggml_cuda_op_silu, true);
}

void ggml_cuda_rms_norm(const ggml_tensor * src0, const ggml_tensor * src1, ggml_tensor * dst) {
    GGML_ASSERT(src0->type == GGML_TYPE_F32 && dst->type == GGML_TYPE_F32);
    ggml_cuda_op(src0, src1, dst, ggml_cuda_op_rms_norm, true);
}

bool ggml_cuda_can_mul_mat(const struct ggml_tensor * src0, const struct ggml_tensor * src1, struct ggml_tensor * dst) {
    GGML_ASSERT(src0->backend != GGML_BACKEND_GPU);
    const int64_t ne10 = src1->ne[0];

    const int64_t ne0 = dst->ne[0];
    const int64_t ne1 = dst->ne[1];

    if (strcmp(dst->name, "KQ") == 0 || strcmp(dst->name, "KQV") == 0) {
        // fprintf(stderr, "(%ld, %ld, %ld, %ld) + (%ld, %ld, %ld, %ld) -> (%ld, %ld, %ld, %ld)\n",
        //         src0->ne[0], src0->ne[1], src0->ne[2], src0->ne[3],
        //         src1->ne[0], src1->ne[1], src1->ne[2], src1->ne[3],
        //         dst->ne[0], dst->ne[1], dst->ne[2], dst->ne[3]);
        return false;
    }

    // TODO: find the optimal values for these
    if ((src0->type == GGML_TYPE_F32 || src0->type == GGML_TYPE_F16 || ggml_is_quantized(src0->type)) &&
        src1->type == GGML_TYPE_F32 &&
        dst->type == GGML_TYPE_F32 &&
        (ne0 >= 32 && ne1 >= 32 && ne10 >= 32)) {
        return true;
    }

    return false;
}

void ggml_cuda_mul_mat(const ggml_tensor * src0, const ggml_tensor * src1, ggml_tensor * dst) {
    if (src0->type == GGML_TYPE_F32) {
        ggml_cuda_op(src0, src1, dst, ggml_cuda_op_mul_mat_cublas, true);
    } else if (ggml_is_quantized(src0->type) || src0->type == GGML_TYPE_F16) {
        if (src1->ne[1] == 1) {
            ggml_cuda_op(src0, src1, dst, ggml_cuda_op_dequantize_mul_mat_vec, false);
        } else {
            ggml_cuda_op(src0, src1, dst, ggml_cuda_op_mul_mat_cublas, true);
        }
    } else {
        GGML_ASSERT(false);
    }
}

void ggml_cuda_load_data(const char * fname, struct ggml_tensor * tensor, const size_t offset, int n_layer) {
    (void) n_layer;
    FILE * fp = fopen(fname, "rb");
    int nrows = ggml_nrows(tensor);
    const size_t nb1 = tensor->nb[1];
    ggml_backend backend = tensor->backend;
    struct ggml_tensor_extra_gpu * extra = (struct ggml_tensor_extra_gpu *) tensor->extra;

    for (int id = 0; id < g_device_count; ++id) {
        extra->data_device[id] = nullptr;

        if (backend == GGML_BACKEND_GPU && id != g_main_device) {
            continue;
        }

        hipSetDevice(id);

        int row_low, row_high;
        if (backend == GGML_BACKEND_GPU) {
            extra->i_device = id;

            row_low = 0;
            row_high = nrows;
        } else if (backend == GGML_BACKEND_GPU_SPLIT) {
            extra->i_device = -1;

            row_low = id == 0 ? 0 : nrows*g_tensor_split[id];
            row_low -= row_low % GGML_CUDA_DMMV_Y;
            row_high = id == g_device_count - 1 ? nrows : nrows*g_tensor_split[id + 1];
            row_high -= row_high % GGML_CUDA_DMMV_Y;
        } else {
            GGML_ASSERT(false);
        }
        if (row_low == row_high) {
            continue;
        }

        int64_t nrows_split = row_high - row_low;

        const size_t offset_split = offset + row_low*nb1;
        const size_t size = ggml_nbytes_split(tensor, nrows_split);

        void * buf;
        CUDA_CHECK(hipMalloc(&buf, size));
        void * buf_host = malloc(size);

#ifdef _WIN32
        int ret = _fseeki64(fp, (__int64) offset_split, SEEK_SET);
#else
        int ret = fseek(fp, (long) offset_split, SEEK_SET);
#endif
        GGML_ASSERT(ret == 0); // same

        size_t ret2 = fread(buf_host, size, 1, fp);
        if (ret2 != 1) {
            fprintf(stderr, "unexpectedly reached end of file");
            exit(1);
        }

        hipMemcpy(buf, buf_host, size, hipMemcpyHostToDevice);
        hipDeviceSynchronize();

        free(buf_host);
        extra->data_device[id] = buf;
    }

    tensor->extra = extra;
    fclose(fp);
}

void ggml_cuda_free_data(struct ggml_tensor * tensor) {
    if (tensor->backend != GGML_BACKEND_GPU && tensor->backend != GGML_BACKEND_GPU_SPLIT) {
        return;
    }

    ggml_tensor_extra_gpu * extra = (ggml_tensor_extra_gpu *) tensor->extra;

    for (int id = 0; id < g_device_count; ++id) {
        if (extra->data_device[id] == nullptr) {
            continue;
        }

        CUDA_CHECK(hipSetDevice(id));
        CUDA_CHECK(hipFree(extra->data_device[id]));
    }

    delete extra;
}

void ggml_cuda_assign_buffers(struct ggml_tensor * tensor, int layer, int n_layer) {
    (void) n_layer;
    const size_t size = ggml_nbytes(tensor);
    GGML_ASSERT(size <= GGML_CUDA_SCRATCH_SIZE);
    if (g_scratch_offset + size > GGML_CUDA_SCRATCH_SIZE) {
        g_scratch_offset = 0;
    }

    tensor->backend = GGML_BACKEND_GPU;
    struct ggml_tensor_extra_gpu * extra = new ggml_tensor_extra_gpu;
    extra->i_device = g_main_device;
    extra->layer = layer;
    struct ggml_tensor_extra_gpu * src0_extra = (ggml_tensor_extra_gpu * ) tensor->src0->extra;

    bool inplace = tensor->src0->data == tensor->data;

    CUDA_CHECK(hipSetDevice(g_main_device));
    if (inplace && tensor->src0->backend == GGML_BACKEND_GPU) {
        extra->data_device[g_main_device] = src0_extra->data_device;
        GGML_ASSERT(false);
    } else {
        char * data = (char *) g_scratch_buffers[g_main_device][g_scratch_index];
        if (data == nullptr) {
            CUDA_CHECK(hipMalloc(&data, GGML_CUDA_SCRATCH_SIZE));
            g_scratch_buffers[g_main_device][g_scratch_index] = data;
        }
        extra->data_device[g_main_device] = data + g_scratch_offset;
    }

    // fprintf(stderr, "data=%p offset=%ld data_device=%p\n", data, g_scratch_offset, extra->data_device[0]);
    g_scratch_offset += size;
    // fprintf(stderr, "%s: scratch %d, %p - %p\n",
    //         tensor->name, g_scratch_index, data + g_scratch_offset, data + g_scratch_offset + size);

    GGML_ASSERT(g_scratch_offset <= GGML_CUDA_SCRATCH_SIZE);
    tensor->extra = extra;
}

void ggml_cuda_set_scratch(int i) {
    if (i == -1) {
        return;
    }
#if false
    fprintf(stderr, "\n%s: switched scratch %d -> %d, old scratch used %.2f MB\n",
            __func__, g_scratch_index, i, g_scratch_offset/1024.0f/1024.0f);
#endif
    g_scratch_index = i;
    g_scratch_offset = 0;
}

bool ggml_cuda_compute_forward(struct ggml_compute_params * params, struct ggml_tensor * tensor){
    ggml_cuda_func_t func;
    const bool any_on_device = tensor->backend == GGML_BACKEND_GPU
        || tensor->src0->backend == GGML_BACKEND_GPU || tensor->src0->backend == GGML_BACKEND_GPU_SPLIT
        || (tensor->src1 != nullptr && tensor->src1->backend == GGML_BACKEND_GPU);

    switch (tensor->op) {
        case GGML_OP_ADD:
            if (!any_on_device) {
                return false;
            }
            func = ggml_cuda_add;
            break;
        case GGML_OP_MUL:
            if (!any_on_device) {
                return false;
            }
            func = ggml_cuda_mul;
            break;
        case GGML_OP_SILU:
            if (!any_on_device) {
                return false;
            }
            func = ggml_cuda_silu;
            break;
        case GGML_OP_RMS_NORM:
            if (!any_on_device) {
                return false;
            }
            func = ggml_cuda_rms_norm;
            break;
        case GGML_OP_MUL_MAT:
            if (!any_on_device && !ggml_cuda_can_mul_mat(tensor->src0, tensor->src1, tensor)) {
                return false;
            }
            func = ggml_cuda_mul_mat;
            break;
        default:
            return false;
    }

    if (params->ith != 0) {
        return true;
    }
    if (params->type == GGML_TASK_INIT || params->type == GGML_TASK_FINALIZE) {
        return true;
    }
    func(tensor->src0, tensor->src1, tensor);
    return true;
}
